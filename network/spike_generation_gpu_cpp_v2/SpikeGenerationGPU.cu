#include <omp.h>
#include <stdio.h>
#include <string.h>

#include <algorithm>
#include <cassert>
#include <iostream>

#include "SpikeGenerationGPU.h"
#include "kernel_spike_generation.h"

SpikeGenerationGPU::SpikeGenerationGPU()
{

};

SpikeGenerationGPU::~SpikeGenerationGPU()
{

};

void SpikeGenerationGPU::entrypoint(
  int64_t input_pointer_addr,
  int64_t input_dim_0,
  int64_t input_dim_1,
  int64_t input_dim_2,
  int64_t input_dim_3,
  int64_t random_values_pointer_addr,
  int64_t random_values_dim_0,
  int64_t random_values_dim_1,
  int64_t random_values_dim_2,
  int64_t random_values_dim_3,
  int64_t output_pointer_addr,
  int64_t output_dim_0,
  int64_t output_dim_1,
  int64_t output_dim_2,
  int64_t output_dim_3,
  int64_t number_of_cpu_processes)
{
  float* input_pointer = (float*)input_pointer_addr;
  float* random_values_pointer = (float*)random_values_pointer_addr;
  int64_t* output_pointer = (int64_t*)output_pointer_addr;

  // Input
  assert((input_pointer != nullptr));
  assert((input_dim_0 > 0));
  assert((input_dim_1 > 0));
  assert((input_dim_2 > 0));
  assert((input_dim_3 > 0));

  // Random
  assert((random_values_pointer != nullptr));
  assert((random_values_dim_0 > 0));
  assert((random_values_dim_1 > 0));
  assert((random_values_dim_2 > 0));
  assert((random_values_dim_3 > 0));

  // Output
  assert((output_pointer != nullptr));
  assert((output_dim_0 > 0));
  assert((output_dim_1 > 0));
  assert((output_dim_2 > 0));
  assert((output_dim_3 > 0));

  // Input
  size_t input_dim_c0 = input_dim_1 * input_dim_2 * input_dim_3;
  size_t input_dim_c1 = input_dim_2 * input_dim_3;
  size_t input_dim_c2 = input_dim_3;

  // Random
  size_t random_values_dim_c0 =
    random_values_dim_1 * random_values_dim_2 * random_values_dim_3;
  size_t random_values_dim_c1 = random_values_dim_2 * random_values_dim_3;
  size_t random_values_dim_c2 = random_values_dim_3;

  // Output
  size_t output_dim_c0 = output_dim_1 * output_dim_2 * output_dim_3;
  size_t output_dim_c1 = output_dim_2 * output_dim_3;
  size_t output_dim_c2 = output_dim_3;

  size_t number_of_pattern = input_dim_0;
  size_t h_dim = input_dim_1;
  size_t spike_dim = output_dim_1;
  size_t x_dim = output_dim_2;
  size_t y_dim = output_dim_2;

  assert((number_of_cpu_processes <= 0));

  gpu_spike_generation(
    input_pointer, input_dim_c0, input_dim_c1, input_dim_c2,
    random_values_pointer, random_values_dim_c0, random_values_dim_c1,
    random_values_dim_c2, output_pointer, output_dim_c0, output_dim_c1,
    output_dim_c2, x_dim, y_dim, spike_dim, h_dim, number_of_pattern);

  return;
};


void SpikeGenerationGPU::gpu_occupancy_measure(
  size_t dim_x,
  size_t dim_y,
  size_t number_of_pattern,
  size_t spike_dim)
{
  grid_and_thread_calculated = false;
  assert((dim_x < 65535));
  assert((dim_y < 65535));

  grid_and_thread_settings.resize(1);

  occupancy_kernel_spike_generation(dim_x, dim_y, number_of_pattern, spike_dim,
    grid_and_thread_settings[0], display_debug);

  grid_and_thread_calculated = true;
  return;
};

void SpikeGenerationGPU::gpu_occupancy_export(
  size_t dim_x,
  size_t dim_y,
  size_t number_of_pattern,
  size_t spike_dim,
  int64_t setting_memory_addr,
  size_t setting_dim_0,
  size_t setting_dim_1)
{
  int64_t* setting_memory = (int64_t*)setting_memory_addr;

  assert((setting_memory != nullptr));
  assert((setting_dim_1 == SPIKE_GENERATION_NUMBER_OF_KERNELS_PARAMETERS));

  gpu_occupancy_measure(dim_x, dim_y, number_of_pattern, spike_dim);
  assert((grid_and_thread_calculated == true));
  assert(
    (grid_and_thread_settings.size() == SPIKE_GENERATION_NUMBER_OF_KERNELS));

  assert((setting_dim_0 == grid_and_thread_settings.size()));

  for (size_t counter_0 = 0; counter_0 < setting_dim_0; counter_0++)
  {
    for (size_t counter_1 = 0; counter_1 < setting_dim_1; counter_1++)
    {
      setting_memory[counter_0 * setting_dim_1 + counter_1] =
        grid_and_thread_settings[counter_0][counter_1];
    }
  }
};

void SpikeGenerationGPU::gpu_occupancy_import(
  int64_t setting_memory_addr,
  size_t setting_dim_0,
  size_t setting_dim_1)
{
  grid_and_thread_calculated = false;

  int64_t* setting_memory = (int64_t*)setting_memory_addr;

  assert((setting_memory != nullptr));
  assert((setting_dim_1 == SPIKE_GENERATION_NUMBER_OF_KERNELS_PARAMETERS));
  assert((setting_dim_0 == SPIKE_GENERATION_NUMBER_OF_KERNELS));

  grid_and_thread_settings.resize(SPIKE_GENERATION_NUMBER_OF_KERNELS);

  for (size_t counter_0 = 0; counter_0 < setting_dim_0; counter_0++)
  {
    grid_and_thread_settings[counter_0].resize(
      SPIKE_GENERATION_NUMBER_OF_KERNELS_PARAMETERS);

    for (size_t counter_1 = 0; counter_1 < setting_dim_1; counter_1++)
    {
      grid_and_thread_settings[counter_0][counter_1] =
        setting_memory[counter_0 * setting_dim_1 + counter_1];
    }
  }

  grid_and_thread_calculated = true;
};

void SpikeGenerationGPU::gpu_spike_generation(
  float* input_pointer,
  size_t input_dim_c0,
  size_t input_dim_c1,
  size_t input_dim_c2,
  float* random_values_pointer,
  size_t random_values_dim_c0,
  size_t random_values_dim_c1,
  size_t random_values_dim_c2,
  int64_t* output_pointer,
  size_t output_dim_c0,
  size_t output_dim_c1,
  size_t output_dim_c2,
  size_t x_dim,
  size_t y_dim,
  size_t spike_dim,
  size_t h_dim,
  size_t number_of_pattern)
{
  if (grid_and_thread_calculated == false)
  {
    gpu_occupancy_measure(x_dim, y_dim, number_of_pattern, spike_dim);
  }
  assert((grid_and_thread_calculated == true));

  hipError_t status;
  assert((x_dim < 65535));
  assert((y_dim < 65535));

  size_t psxy_block_dim_c0 = spike_dim * x_dim * y_dim;
  size_t psxy_block_dim_c1 = x_dim * y_dim;
  size_t psxy_block_dim_c2 = y_dim;

  kernel_spike_generation<<<
    dim3(grid_and_thread_settings[0][0], grid_and_thread_settings[0][1],
      grid_and_thread_settings[0][2]),
    dim3(grid_and_thread_settings[0][3], grid_and_thread_settings[0][4],
      grid_and_thread_settings[0][5])>>>(
        input_pointer, input_dim_c0, input_dim_c1, input_dim_c2,
        random_values_pointer, random_values_dim_c0, random_values_dim_c1,
        random_values_dim_c2, output_pointer, output_dim_c0, output_dim_c1,
        output_dim_c2, x_dim, y_dim, spike_dim, h_dim, psxy_block_dim_c0,
        psxy_block_dim_c1, psxy_block_dim_c2, grid_and_thread_settings[0][6]);

  status = hipDeviceSynchronize();
  assert((status == hipSuccess));

  return;
};