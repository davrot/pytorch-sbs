#include "hip/hip_runtime.h"
#include "SpikeGenerationGPU.h"

#include <omp.h>
#include <stdio.h>
#include <string.h>

#include <algorithm>
#include <cassert>
#include <iostream>


SpikeGenerationGPU::SpikeGenerationGPU()
{

};

SpikeGenerationGPU::~SpikeGenerationGPU()
{

};

void SpikeGenerationGPU::entrypoint(
    int64_t input_pointer_addr,
    int64_t input_dim_0,
    int64_t input_dim_1,
    int64_t input_dim_2,
    int64_t input_dim_3,
    int64_t random_values_pointer_addr,
    int64_t random_values_dim_0,
    int64_t random_values_dim_1,
    int64_t random_values_dim_2,
    int64_t random_values_dim_3,
    int64_t output_pointer_addr,
    int64_t output_dim_0,
    int64_t output_dim_1,
    int64_t output_dim_2,
    int64_t output_dim_3,
    int64_t number_of_cpu_processes)
{

    float* input_pointer = (float*)input_pointer_addr;
    float* random_values_pointer = (float*)random_values_pointer_addr;
    int64_t* output_pointer = (int64_t*)output_pointer_addr;

    // Input
    assert((input_pointer != nullptr));
    assert((input_dim_0 > 0));
    assert((input_dim_1 > 0));
    assert((input_dim_2 > 0));
    assert((input_dim_3 > 0));

    // Random
    assert((random_values_pointer != nullptr));
    assert((random_values_dim_0 > 0));
    assert((random_values_dim_1 > 0));
    assert((random_values_dim_2 > 0));
    assert((random_values_dim_3 > 0));

    // Output
    assert((output_pointer != nullptr));
    assert((output_dim_0 > 0));
    assert((output_dim_1 > 0));
    assert((output_dim_2 > 0));
    assert((output_dim_3 > 0));

    // Input
    size_t input_dim_c0 = input_dim_1 * input_dim_2 * input_dim_3;
    size_t input_dim_c1 = input_dim_2 * input_dim_3;
    size_t input_dim_c2 = input_dim_3;

    // Random
    size_t random_values_dim_c0 =
        random_values_dim_1 * random_values_dim_2 * random_values_dim_3;
    size_t random_values_dim_c1 =
        random_values_dim_2 * random_values_dim_3;
    size_t random_values_dim_c2 = random_values_dim_3;

    // Output
    size_t output_dim_c0 =
        output_dim_1 * output_dim_2 * output_dim_3;
    size_t output_dim_c1 = output_dim_2 * output_dim_3;
    size_t output_dim_c2 = output_dim_3;

    size_t number_of_pattern = input_dim_0;
    size_t h_dim = input_dim_1;
    size_t spike_dim = output_dim_1;
    size_t x_dim = output_dim_2;
    size_t y_dim = output_dim_2;

    assert((number_of_cpu_processes <= 0));

    gpu_spike_generation(
        input_pointer,
        input_dim_c0,
        input_dim_c1,
        input_dim_c2,
        random_values_pointer,
        random_values_dim_c0,
        random_values_dim_c1,
        random_values_dim_c2,
        output_pointer,
        output_dim_c0,
        output_dim_c1,
        output_dim_c2,
        x_dim,
        y_dim,
        spike_dim,
        h_dim,
        number_of_pattern);

    return;
};

__device__ size_t gpu_lower_bound(float* __restrict__ data_ptr,
    size_t data_length,
    size_t data_ptr_stride,
    float compare_to_value)
{

    size_t start_of_range = 0;
    size_t length_of_range = data_length;

    while (length_of_range != 0)
    {
        size_t half_length = length_of_range >> 1;
        size_t actual_position = start_of_range + half_length;

        if (data_ptr[actual_position * data_ptr_stride] < compare_to_value)
        {
            start_of_range = ++actual_position;
            length_of_range -= half_length + 1;
        }
        else
            length_of_range = half_length;
    }
    return start_of_range;
};

__global__ void kernel_spike_generation(
    float* __restrict__ input_pointer,
    size_t input_dim_c0,
    size_t input_dim_c1,
    size_t input_dim_c2,
    float* __restrict__ random_values_pointer,
    size_t random_values_dim_c0,
    size_t random_values_dim_c1,
    size_t random_values_dim_c2,
    int64_t* __restrict__ output_pointer,
    size_t output_dim_c0,
    size_t output_dim_c1,
    size_t output_dim_c2,
    size_t x_dim,
    size_t y_dim,
    size_t spike_dim,
    size_t h_dim,
    size_t max_threadable_tasks)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < max_threadable_tasks)
    {

        size_t pattern_id = idx / spike_dim;
        size_t position_spike = idx - (pattern_id * spike_dim);

        size_t position_x = blockIdx.y;
        size_t position_y = blockIdx.z;

        float* p_ptr = input_pointer + pattern_id * input_dim_c0 +
            position_x * input_dim_c2 + position_y;

        int64_t* out_ptr = output_pointer + pattern_id * output_dim_c0 +
            position_x * output_dim_c2 + position_y
            + position_spike * output_dim_c1;

        float* rand_ptr = random_values_pointer +
            pattern_id * random_values_dim_c0 +
            position_x * random_values_dim_c2 + position_y
            + position_spike * random_values_dim_c1;

        *out_ptr = gpu_lower_bound(p_ptr,
            h_dim,
            input_dim_c1,
            *rand_ptr);
    }
};

void SpikeGenerationGPU::gpu_spike_generation(
    float* input_pointer,
    size_t input_dim_c0,
    size_t input_dim_c1,
    size_t input_dim_c2,
    float* random_values_pointer,
    size_t random_values_dim_c0,
    size_t random_values_dim_c1,
    size_t random_values_dim_c2,
    int64_t* output_pointer,
    size_t output_dim_c0,
    size_t output_dim_c1,
    size_t output_dim_c2,
    size_t x_dim,
    size_t y_dim,
    size_t spike_dim,
    size_t h_dim,
    size_t number_of_pattern)
{
    hipError_t status;
    assert((x_dim < 65535));
    assert((y_dim < 65535));

    // //////////////////////////////////////
    // Calculate the distribution on the GPU
    // //////////////////////////////////////

    int min_grid_size;
    int block_size;
    int grid_size;

    size_t dynamic_s_mem_size = 0;
    size_t max_threadable_tasks = number_of_pattern * spike_dim * x_dim * y_dim;

    status = hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
        (void*)kernel_spike_generation,
        dynamic_s_mem_size, max_threadable_tasks);
    if (status != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: "
            << __FILE__
            << ":"
            << __LINE__
            << std::endl;
        std::cerr << hipGetErrorString(status) << std::endl;
    }
    assert((status == hipSuccess));

    grid_size = ((number_of_pattern * spike_dim) + block_size - 1) / block_size;

    dim3 grid(grid_size, x_dim, y_dim);


    kernel_spike_generation<<<grid, block_size >>>(
        input_pointer,
        input_dim_c0,
        input_dim_c1,
        input_dim_c2,
        random_values_pointer,
        random_values_dim_c0,
        random_values_dim_c1,
        random_values_dim_c2,
        output_pointer,
        output_dim_c0,
        output_dim_c1,
        output_dim_c2,
        x_dim,
        y_dim,
        spike_dim,
        h_dim,
        (number_of_pattern * spike_dim));

    status = hipDeviceSynchronize();
    if (status != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: "
            << __FILE__
            << ":"
            << __LINE__
            << std::endl;
        std::cerr << hipGetErrorString(status) << std::endl;
    }
    assert((status == hipSuccess));
    return;
};


void SpikeGenerationGPU::gpu_occupancy_export(
    size_t dim_x,
    size_t dim_y,
    size_t number_of_pattern,
    size_t spike_dim,
    int64_t setting_memory_addr,
    size_t setting_dim_0,
    size_t setting_dim_1)
{
    return;
};

void SpikeGenerationGPU::gpu_occupancy_import(
    int64_t setting_memory_addr,
    size_t setting_dim_0,
    size_t setting_dim_1)
{
    return;
};