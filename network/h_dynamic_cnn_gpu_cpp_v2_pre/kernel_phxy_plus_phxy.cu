#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>

#include "kernel_helper_functions.h"
#include "kernel_phxy_plus_phxy.h"

__global__ void kernel_phxy_plus_phxy(float* __restrict__ phxy_memory_a,
                                      float* __restrict__ phxy_memory_b,
                                      size_t max_idx) {
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < max_idx) {
    phxy_memory_a[idx] += phxy_memory_b[idx];
  }
};

void occupancy_kernel_phxy_plus_phxy(size_t dim_x, size_t dim_y,
                                     size_t number_of_pattern, size_t h_dim,
                                     std::vector<size_t>& output,
                                     bool display_debug) {
  size_t max_threadable_tasks;
  hipError_t status;

  int min_grid_size;
  int thread_block_size;
  int grid_size;

  max_threadable_tasks = number_of_pattern * h_dim * dim_x * dim_y;

  status = hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, (void*)kernel_phxy_plus_phxy, 0,
      max_threadable_tasks);
  assert((status == hipSuccess));

  grid_size =
      (max_threadable_tasks + thread_block_size - 1) / thread_block_size;

  output.resize(7);
  output[0] = grid_size;
  output[1] = 1;
  output[2] = 1;
  output[3] = thread_block_size;
  output[4] = 1;
  output[5] = 1;
  output[6] = max_threadable_tasks;

  if (display_debug == true) {
    std::cout << "kernel_phxy_plus_phxy:" << std::endl;
    kernel_debug_plot(output, display_debug);
  }
};