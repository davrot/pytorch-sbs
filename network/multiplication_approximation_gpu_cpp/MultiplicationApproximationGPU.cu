#include "hip/hip_runtime.h"
#include "MultiplicationApproximationGPU.h"

#include <omp.h>
#include <stdio.h>
#include <string.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <iostream>
#include <vector>

#include "gpu_approximation_multiplication_function.cu"

MultiplicationApproximationGPU::MultiplicationApproximationGPU()
{

};

MultiplicationApproximationGPU::~MultiplicationApproximationGPU()
{

};

void MultiplicationApproximationGPU::entrypoint(
    int64_t np_input_pointer_addr, 
    int64_t np_weight_pointer_addr,
    int64_t np_output_pointer_addr, 
    int64_t pattern_dim, 
    int64_t feature_dim,
    int64_t x_dim, 
    int64_t y_dim, 
    int64_t input_channel_dim,
    int64_t number_of_processes, 
    bool approximation_enable,
    int64_t number_of_trunc_bits, 
    int64_t number_of_frac)
{

    // int64_t number_of_pattern = pattern_dim;

    float* np_input_pointer = (float*)np_input_pointer_addr;
    float* np_weight_pointer = (float*)np_weight_pointer_addr;
    float* np_output_pointer = (float*)np_output_pointer_addr;

    assert((np_input_pointer != nullptr));
    assert((np_output_pointer != nullptr));
    assert((np_weight_pointer != nullptr));

    assert((pattern_dim > 0));
    assert((feature_dim > 0));
    assert((x_dim > 0));
    assert((y_dim > 0));
    assert((input_channel_dim > 0));

    assert ((number_of_processes <= 0));

    calculate_gpu(np_input_pointer, np_weight_pointer,
        np_output_pointer, pattern_dim, feature_dim, x_dim, y_dim,
        input_channel_dim, approximation_enable,
        number_of_trunc_bits, number_of_frac);

    return;
};

__global__ void kernel_approx_multiplication(
    float* __restrict__ input_pointer, 
    float* __restrict__ weight_pointer,
    float* __restrict__ output_pointer, 
    uint64_t pattern_dim,
    uint64_t feature_dim, 
    uint64_t x_dim, 
    uint64_t y_dim,
    uint64_t input_channel_dim, 
    size_t max_threadable_tasks,
    uint64_t input_index_scale, 
    uint64_t number_of_frac_bits,
    bool approximation_enable, 
    uint64_t number_of_trunc_bits,
    uint32_t ap_mask)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < max_threadable_tasks)
    {
        int pattern_id = idx / feature_dim;
        int feature_id = idx - (pattern_id * feature_dim);
        int x_id = blockIdx.y;
        int y_id = blockIdx.z;

        float* weight_pointer_sub = weight_pointer + feature_id * input_channel_dim;
        float* input_pointer_sub = input_pointer + pattern_id * input_channel_dim * x_dim * y_dim + x_id * y_dim + y_id;
        float* output_pointer_sub = output_pointer +
            pattern_id * feature_dim * x_dim * y_dim +
            feature_id * x_dim * y_dim + x_id * y_dim + y_id;
        *output_pointer_sub = 0.0;

        for (size_t counter = 0; counter < input_channel_dim; counter++)
        {
            *output_pointer_sub += gpu_approximation_multiplication_function(
                weight_pointer_sub[counter],
                input_pointer_sub[counter * input_index_scale],
                number_of_frac_bits, approximation_enable,
                number_of_trunc_bits, ap_mask);
        }
    }
};

void MultiplicationApproximationGPU::calculate_gpu(
    float* np_input_pointer,
    float* np_weight_pointer,
    float* np_output_pointer, 
    size_t pattern_dim,
    size_t feature_dim, 
    size_t x_dim, 
    size_t y_dim,
    size_t input_channel_dim,
    bool approximation_enable, 
    size_t number_of_trunc_bits,
    size_t number_of_frac_bits)
{

    uint32_t ap_mask = static_cast<uint64_t>(pow(2, number_of_trunc_bits)) - 1;
    // std::cout << approximation_enable << std::endl;
    // std::cout << number_of_trunc_bits << std::endl;
    // std::cout << number_of_frac_bits << std::endl;

    hipError_t status;
    assert((x_dim < 65535));
    assert((y_dim < 65535));

    // //////////////////////////////////////
    // Calculate the distribution on the GPU
    // //////////////////////////////////////

    int min_grid_size;
    int block_size;
    int grid_size;

    size_t dynamic_s_mem_size = 0;
    size_t max_threadable_tasks = pattern_dim * feature_dim * x_dim * y_dim;

    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=blocksize#occupancy-calculator
    status = hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
        (void*)kernel_approx_multiplication,
        dynamic_s_mem_size, max_threadable_tasks);
    assert((status == hipSuccess));

    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications
    // Maximum dimensionality of grid of thread blocks: 3
    // Maximum x -dimension of a grid of thread blocks: (2^31)-1
    // Maximum y- or z-dimension of a grid of thread blocks: 65535

    // Round up according to array size
    grid_size = ((pattern_dim * feature_dim) + block_size - 1) / block_size;

    // std::cout << min_grid_size << std::endl;
    // std::cout << grid_size << std::endl;
    // std::cout << block_size << std::endl;
    // std::cout << max_threadable_tasks << std::endl;

    dim3 grid(grid_size, x_dim, y_dim);

    kernel_approx_multiplication<<<grid, block_size>>>(np_input_pointer,
        np_weight_pointer,
        np_output_pointer,
        pattern_dim,
        feature_dim,
        x_dim,
        y_dim,
        input_channel_dim,
        (pattern_dim * feature_dim),
        (x_dim * y_dim),
        number_of_frac_bits,
        approximation_enable,
        number_of_trunc_bits,
        ap_mask);

    status = hipDeviceSynchronize();
    assert((status == hipSuccess));
    return;
};
