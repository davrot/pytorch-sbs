#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>

#include "kernel_helper_functions.h"
#include "kernel_pxy_reciprocal.h"

__global__ void kernel_pxy_reciprocal(float* __restrict__ pxy_memory,
                                      size_t max_idx) {
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < max_idx) {
    pxy_memory[idx] = 1.0 / pxy_memory[idx];
  }
};

void occupancy_kernel_pxy_reciprocal(size_t dim_x, size_t dim_y,
                                     size_t number_of_pattern, size_t h_dim,
                                     std::vector<size_t>& output,
                                     bool display_debug) {
  size_t max_threadable_tasks;
  hipError_t status;

  int min_grid_size;
  int thread_block_size;
  int grid_size;

  max_threadable_tasks = number_of_pattern * dim_x * dim_y;

  status = hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, (void*)kernel_pxy_reciprocal, 0,
      max_threadable_tasks);
  assert((status == hipSuccess));

  grid_size =
      (max_threadable_tasks + thread_block_size - 1) / thread_block_size;

  output.resize(7);
  output[0] = grid_size;
  output[1] = 1;
  output[2] = 1;
  output[3] = thread_block_size;
  output[4] = 1;
  output[5] = 1;
  output[6] = max_threadable_tasks;

  if (display_debug == true) {
    std::cout << "kernel_pxy_reciprocal:" << std::endl;
    kernel_debug_plot(output, display_debug);
  }
};