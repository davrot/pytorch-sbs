#include <omp.h>
#include <stdio.h>
#include <string.h>

#include <algorithm>
#include <cassert>
#include <iostream>

#include "SpikeGeneration2DManyIP.h"
#include "kernel_spike_generation.h"

SpikeGeneration2DManyIP::SpikeGeneration2DManyIP(){

};

SpikeGeneration2DManyIP::~SpikeGeneration2DManyIP(){

};

bool SpikeGeneration2DManyIP::spike_generation_entrypoint(
    int64_t input_pointer_addr, int64_t input_dim_0, int64_t input_dim_1,
    int64_t input_dim_2, int64_t input_dim_3,
    int64_t random_values_pointer_addr, int64_t random_values_dim_0,
    int64_t random_values_dim_1, int64_t random_values_dim_2,
    int64_t random_values_dim_3, int64_t output_pointer_addr,
    int64_t output_dim_0, int64_t output_dim_1, int64_t output_dim_2,
    int64_t output_dim_3, int64_t number_of_cpu_processes) {
  float* input_pointer = (float*)input_pointer_addr;
  float* random_values_pointer = (float*)random_values_pointer_addr;
  int64_t* output_pointer = (int64_t*)output_pointer_addr;

  // Input
  assert((input_pointer != nullptr));
  assert((input_dim_0 > 0));
  assert((input_dim_1 > 0));
  assert((input_dim_2 > 0));
  assert((input_dim_3 > 0));

  // Random
  assert((random_values_pointer != nullptr));
  assert((random_values_dim_0 > 0));
  assert((random_values_dim_1 > 0));
  assert((random_values_dim_2 > 0));
  assert((random_values_dim_3 > 0));

  // Output
  assert((output_pointer != nullptr));
  assert((output_dim_0 > 0));
  assert((output_dim_1 > 0));
  assert((output_dim_2 > 0));
  assert((output_dim_3 > 0));

  // Input
  size_t input_dim_c0 = input_dim_1 * input_dim_2 * input_dim_3;
  size_t input_dim_c1 = input_dim_2 * input_dim_3;
  size_t input_dim_c2 = input_dim_3;

  // Random
  size_t random_values_dim_c0 =
      random_values_dim_1 * random_values_dim_2 * random_values_dim_3;
  size_t random_values_dim_c1 = random_values_dim_2 * random_values_dim_3;
  size_t random_values_dim_c2 = random_values_dim_3;

  // Output
  size_t output_dim_c0 = output_dim_1 * output_dim_2 * output_dim_3;
  size_t output_dim_c1 = output_dim_2 * output_dim_3;
  size_t output_dim_c2 = output_dim_3;

  size_t number_of_pattern = input_dim_0;
  size_t h_dim = input_dim_1;
  size_t spike_dim = output_dim_1;
  size_t x_dim = output_dim_2;
  size_t y_dim = output_dim_2;

  if (number_of_cpu_processes > 0) {
    omp_set_num_threads(number_of_cpu_processes);
    // DEBUG:
    // omp_set_num_threads(1);

    size_t pattern_id;

#pragma omp parallel for
    for (pattern_id = 0; pattern_id < number_of_pattern; pattern_id++) {
      spike_generation(
          input_pointer, input_dim_c0, input_dim_c1, input_dim_c2,
          random_values_pointer, random_values_dim_c0, random_values_dim_c1,
          random_values_dim_c2, output_pointer, output_dim_c0, output_dim_c1,
          output_dim_c2, x_dim, y_dim, spike_dim, h_dim, pattern_id);
    }
  } else {
    gpu_spike_generation(
        input_pointer, input_dim_c0, input_dim_c1, input_dim_c2,
        random_values_pointer, random_values_dim_c0, random_values_dim_c1,
        random_values_dim_c2, output_pointer, output_dim_c0, output_dim_c1,
        output_dim_c2, x_dim, y_dim, spike_dim, h_dim, number_of_pattern);
  }

  return true;
};

bool SpikeGeneration2DManyIP::spike_generation(
    float* input_pointer, size_t input_dim_c0, size_t input_dim_c1,
    size_t input_dim_c2, float* random_values_pointer,
    size_t random_values_dim_c0, size_t random_values_dim_c1,
    size_t random_values_dim_c2, int64_t* output_pointer, size_t output_dim_c0,
    size_t output_dim_c1, size_t output_dim_c2, size_t x_dim, size_t y_dim,
    size_t spike_dim, size_t h_dim, size_t pattern_id) {
  size_t counter;
  size_t counter_x = 0;
  size_t counter_y = 0;

  float* p_ptr = nullptr;
  int64_t* out_ptr = nullptr;
  float* rand_ptr = nullptr;

  for (counter_x = 0; counter_x < x_dim; counter_x++) {
    for (counter_y = 0; counter_y < y_dim; counter_y++) {
      p_ptr = input_pointer + pattern_id * input_dim_c0 +
              counter_x * input_dim_c2 + counter_y;
      // + counter * input_dim_c1

      out_ptr = output_pointer + pattern_id * output_dim_c0 +
                counter_x * output_dim_c2 + counter_y;
      // + counter * output_dim_c1

      rand_ptr = random_values_pointer + pattern_id * random_values_dim_c0 +
                 counter_x * random_values_dim_c2 + counter_y;
      // + counter * random_values_dim_c1

      for (counter = 0; counter < spike_dim; counter++) {
        out_ptr[counter * output_dim_c1] =
            lower_bound(p_ptr, h_dim, input_dim_c1,
                        rand_ptr[counter * random_values_dim_c1]);
      }
    }
  }

  return true;
};

// algorithmic idea stolen from libc++
size_t SpikeGeneration2DManyIP::lower_bound(float* data_ptr, size_t data_length,
                                            size_t data_ptr_stride,
                                            float compare_to_value) {
  size_t start_of_range = 0;
  size_t length_of_range = data_length;

  while (length_of_range != 0) {
    size_t half_length = length_of_range >> 1;
    size_t actual_position = start_of_range + half_length;

    if (data_ptr[actual_position * data_ptr_stride] < compare_to_value) {
      start_of_range = ++actual_position;
      length_of_range -= half_length + 1;
    } else
      length_of_range = half_length;
  }
  return start_of_range;
};

void SpikeGeneration2DManyIP::gpu_occupancy_measure(size_t dim_x, size_t dim_y,
                                                    size_t number_of_pattern,
                                                    size_t spike_dim) {
  grid_and_thread_calculated = false;
  assert((dim_x < 65535));
  assert((dim_y < 65535));

  grid_and_thread_settings.resize(1);

  occupancy_kernel_spike_generation(dim_x, dim_y, number_of_pattern, spike_dim,
                                    grid_and_thread_settings[0], display_debug);

  grid_and_thread_calculated = true;
  return;
};

void SpikeGeneration2DManyIP::gpu_occupancy_export(
    size_t dim_x, size_t dim_y, size_t number_of_pattern, size_t spike_dim,
    int64_t setting_memory_addr, size_t setting_dim_0, size_t setting_dim_1) {
  int64_t* setting_memory = (int64_t*)setting_memory_addr;

  assert((setting_memory != nullptr));
  assert((setting_dim_1 == SPIKE_GENERATION_NUMBER_OF_KERNELS_PARAMETERS));

  gpu_occupancy_measure(dim_x, dim_y, number_of_pattern, spike_dim);
  assert((grid_and_thread_calculated == true));
  assert(
      (grid_and_thread_settings.size() == SPIKE_GENERATION_NUMBER_OF_KERNELS));

  assert((setting_dim_0 == grid_and_thread_settings.size()));

  for (size_t counter_0 = 0; counter_0 < setting_dim_0; counter_0++) {
    for (size_t counter_1 = 0; counter_1 < setting_dim_1; counter_1++) {
      setting_memory[counter_0 * setting_dim_1 + counter_1] =
          grid_and_thread_settings[counter_0][counter_1];
    }
  }
};

void SpikeGeneration2DManyIP::gpu_occupancy_import(int64_t setting_memory_addr,
                                                   size_t setting_dim_0,
                                                   size_t setting_dim_1) {
  grid_and_thread_calculated = false;

  int64_t* setting_memory = (int64_t*)setting_memory_addr;

  assert((setting_memory != nullptr));
  assert((setting_dim_1 == SPIKE_GENERATION_NUMBER_OF_KERNELS_PARAMETERS));
  assert((setting_dim_0 == SPIKE_GENERATION_NUMBER_OF_KERNELS));

  grid_and_thread_settings.resize(SPIKE_GENERATION_NUMBER_OF_KERNELS);

  for (size_t counter_0 = 0; counter_0 < setting_dim_0; counter_0++) {
    grid_and_thread_settings[counter_0].resize(
        SPIKE_GENERATION_NUMBER_OF_KERNELS_PARAMETERS);

    for (size_t counter_1 = 0; counter_1 < setting_dim_1; counter_1++) {
      grid_and_thread_settings[counter_0][counter_1] =
          setting_memory[counter_0 * setting_dim_1 + counter_1];
    }
  }

  grid_and_thread_calculated = true;
};

bool SpikeGeneration2DManyIP::gpu_spike_generation(
    float* input_pointer, size_t input_dim_c0, size_t input_dim_c1,
    size_t input_dim_c2, float* random_values_pointer,
    size_t random_values_dim_c0, size_t random_values_dim_c1,
    size_t random_values_dim_c2, int64_t* output_pointer, size_t output_dim_c0,
    size_t output_dim_c1, size_t output_dim_c2, size_t x_dim, size_t y_dim,
    size_t spike_dim, size_t h_dim, size_t number_of_pattern) {
  if (grid_and_thread_calculated == false) {
    gpu_occupancy_measure(x_dim, y_dim, number_of_pattern, spike_dim);
  }
  assert((grid_and_thread_calculated == true));

  hipError_t status;
  assert((x_dim < 65535));
  assert((y_dim < 65535));

  size_t psxy_block_dim_c0 = spike_dim * x_dim * y_dim;
  size_t psxy_block_dim_c1 = x_dim * y_dim;
  size_t psxy_block_dim_c2 = y_dim;

  kernel_spike_generation<<<
      dim3(grid_and_thread_settings[0][0], grid_and_thread_settings[0][1],
           grid_and_thread_settings[0][2]),
      dim3(grid_and_thread_settings[0][3], grid_and_thread_settings[0][4],
           grid_and_thread_settings[0][5])>>>(
      input_pointer, input_dim_c0, input_dim_c1, input_dim_c2,
      random_values_pointer, random_values_dim_c0, random_values_dim_c1,
      random_values_dim_c2, output_pointer, output_dim_c0, output_dim_c1,
      output_dim_c2, x_dim, y_dim, spike_dim, h_dim, psxy_block_dim_c0,
      psxy_block_dim_c1, psxy_block_dim_c2, grid_and_thread_settings[0][6]);

  status = hipDeviceSynchronize();
  assert((status == hipSuccess));

  return true;
};