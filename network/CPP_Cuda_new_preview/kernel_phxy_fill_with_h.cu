#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>

#include "kernel_helper_functions.h"
#include "kernel_phxy_fill_with_h.h"

__global__ void kernel_phxy_fill_with_h(float* __restrict__ h_memory,
                                        float* __restrict__ phxy_memory,
                                        size_t phxy_dim_c0, size_t phxy_dim_c1,
                                        size_t phxy_dim_c2, size_t h_dim,
                                        size_t block_dim_c0,
                                        size_t block_dim_c1,
                                        size_t block_dim_c2, size_t max_idx) {
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < max_idx) {
    size_t pattern_id = idx / block_dim_c0;
    idx -= pattern_id * block_dim_c0;
    size_t idx_h = idx / block_dim_c1;
    idx -= idx_h * block_dim_c1;
    size_t position_x = idx / block_dim_c2;
    idx -= position_x * block_dim_c2;
    size_t position_y = idx;

    phxy_memory[pattern_id * phxy_dim_c0 + idx_h * phxy_dim_c1 +
                position_x * phxy_dim_c2 + position_y] = h_memory[idx_h];
  }
};

void occupancy_kernel_phxy_fill_with_h(size_t dim_x, size_t dim_y,
                                       size_t number_of_pattern, size_t h_dim,
                                       std::vector<size_t>& output,
                                       bool display_debug) {
  size_t max_threadable_tasks;
  hipError_t status;

  int min_grid_size;
  int thread_block_size;
  int grid_size;

  max_threadable_tasks = number_of_pattern * h_dim * dim_x * dim_y;

  status = hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size, (void*)kernel_phxy_fill_with_h, 0,
      max_threadable_tasks);
  assert((status == hipSuccess));

  grid_size =
      (max_threadable_tasks + thread_block_size - 1) / thread_block_size;

  output.resize(7);
  output[0] = grid_size;
  output[1] = 1;
  output[2] = 1;
  output[3] = thread_block_size;
  output[4] = 1;
  output[5] = 1;
  output[6] = max_threadable_tasks;

  if (display_debug == true) {
    std::cout << "kernel_phxy_fill_with_h:" << std::endl;
    kernel_debug_plot(output, display_debug);
  }
};