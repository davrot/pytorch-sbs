#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <string.h>

#include <algorithm>
#include <cassert>
#include <iostream>

#include "HDynamicCNNManyIP.h"
#include "approximation_multiplication_function.h"
#include "kernel_approximation_multiplication.h"
#include "kernel_phxy_fill_with_h.h"
#include "kernel_phxy_fill_with_spike_selected_w.h"
#include "kernel_phxy_one_over_sum_into_pxy.h"
#include "kernel_phxy_plus_phxy.h"
#include "kernel_phxy_plus_pxy.h"
#include "kernel_phxy_times_phxy_equals_phxy.h"
#include "kernel_phxy_times_pxy.h"
#include "kernel_pxy_plus_v.h"
#include "kernel_pxy_reciprocal.h"
#include "kernel_pxy_set_to_v.h"
#include "kernel_pxy_time_pxy.h"
#include "kernel_pxy_times_spike_selected_sxy.h"
#include "kernel_pxy_times_v.h"

HDynamicCNNManyIP::HDynamicCNNManyIP(){

};

HDynamicCNNManyIP::~HDynamicCNNManyIP(){

};

bool HDynamicCNNManyIP::update_entrypoint(
    int64_t h_pointer_addr, int64_t h_dim_0, int64_t h_dim_1, int64_t h_dim_2,
    int64_t h_dim_3, int64_t epsilon_xy_pointer_addr, int64_t epsilon_xy_dim_0,
    int64_t epsilon_xy_dim_1, int64_t epsilon_xy_dim_2,
    int64_t epsilon_t_pointer_addr, int64_t epsilon_t_dim_0,
    int64_t weights_pointer_addr, int64_t weights_dim_0, int64_t weights_dim_1,
    int64_t input_pointer_addr, int64_t input_dim_0, int64_t input_dim_1,
    int64_t input_dim_2, int64_t input_dim_3, int64_t init_vector_pointer_addr,
    int64_t init_vector_dim_0, int64_t number_of_processes,
    float forgetting_offset, int64_t gpu_tuning_factor
    // ,bool approximation_multiplication_enable, uint64_t
    // number_of_frac_bits, bool approximation_enable,
    // uint64_t number_of_trunc_bits
) {
  bool approximation_multiplication_enable = false;
  uint64_t number_of_frac_bits = 1;
  bool approximation_enable = false;
  uint64_t number_of_trunc_bits = false;

  uint32_t ap_mask = static_cast<uint64_t>(pow(2, number_of_trunc_bits)) - 1;

  size_t number_of_pattern = input_dim_0;

  size_t h_dim = init_vector_dim_0;
  float* h_init_ptr = (float*)init_vector_pointer_addr;
  assert((h_init_ptr != nullptr));
  assert((h_dim > 0));

  float* h_pointer = (float*)h_pointer_addr;
  assert((h_pointer != nullptr));
  assert((h_dim_0 > 0));
  assert((h_dim_1 > 0));
  assert((h_dim_2 > 0));
  assert((h_dim_3 > 0));

  size_t h_dim_c0 = h_dim_1 * h_dim_2 * h_dim_3;
  size_t h_dim_c1 = h_dim_2 * h_dim_3;
  size_t h_dim_c2 = h_dim_3;

  float* epsilon_xy_pointer = (float*)epsilon_xy_pointer_addr;
  assert((epsilon_xy_pointer != nullptr));
  assert((epsilon_xy_dim_0 > 0));
  assert((epsilon_xy_dim_1 > 0));

  size_t epsilon_xy_dim_c0 = epsilon_xy_dim_2 * epsilon_xy_dim_1;
  size_t epsilon_xy_dim_c1 = epsilon_xy_dim_2;

  float* epsilon_t_pointer = (float*)epsilon_t_pointer_addr;
  assert((epsilon_t_pointer != nullptr));
  assert((epsilon_t_dim_0 > 0));

  float* weights_pointer = (float*)weights_pointer_addr;
  assert((weights_pointer != nullptr));
  assert((weights_dim_0 > 0));
  assert((weights_dim_1 > 0));

  size_t weights_dim_c0 = weights_dim_1;

  int64_t* input_pointer = (int64_t*)input_pointer_addr;
  assert((input_pointer != nullptr));
  assert((input_dim_0 > 0));
  assert((input_dim_1 > 0));
  assert((input_dim_2 > 0));
  assert((input_dim_3 > 0));

  size_t input_dim_c0 = input_dim_1 * input_dim_2 * input_dim_3;
  size_t input_dim_c1 = input_dim_2 * input_dim_3;
  size_t input_dim_c2 = input_dim_3;

  assert((h_dim == weights_dim_1));
  size_t number_of_spikes = input_dim_1;
  size_t dim_x = input_dim_2;
  size_t dim_y = input_dim_3;

  float forgetting_offset_local = forgetting_offset / static_cast<float>(h_dim);

  // --------------------
  if (number_of_processes > 0) {
    omp_set_num_threads(number_of_processes);

    size_t pattern_id;
#pragma omp parallel for
    for (pattern_id = 0; pattern_id < number_of_pattern; pattern_id++) {
      update(h_init_ptr, h_pointer, h_dim_c0, h_dim_c1, h_dim_c2, h_dim,
             epsilon_xy_pointer, epsilon_xy_dim_c0, epsilon_xy_dim_c1,
             epsilon_t_pointer, weights_pointer, weights_dim_c0, input_pointer,
             input_dim_c0, input_dim_c1, input_dim_c2, number_of_spikes, dim_x,
             dim_y, forgetting_offset, forgetting_offset_local, pattern_id,
             approximation_multiplication_enable, number_of_frac_bits,
             approximation_enable, number_of_trunc_bits, ap_mask);
    }
  } else {
    gpu_update(h_init_ptr, h_pointer, h_dim_c0, h_dim_c1, h_dim_c2, h_dim,
               epsilon_xy_pointer, epsilon_xy_dim_c0, epsilon_xy_dim_c1,
               epsilon_t_pointer, weights_pointer, weights_dim_c0,
               input_pointer, input_dim_c0, input_dim_c1, input_dim_c2,
               number_of_spikes, dim_x, dim_y, forgetting_offset,
               forgetting_offset_local, number_of_pattern, gpu_tuning_factor,
               approximation_multiplication_enable, number_of_frac_bits,
               approximation_enable, number_of_trunc_bits, ap_mask);
  }
  return true;
};

bool HDynamicCNNManyIP::update(
    float* h_init_ptr, float* h_pointer, size_t h_dim_c0, size_t h_dim_c1,
    size_t h_dim_c2, size_t h_dim, float* epsilon_xy_pointer,
    size_t epsilon_xy_dim_c0, size_t epsilon_xy_dim_c1,
    float* epsilon_t_pointer, float* weights_pointer, size_t weights_dim_c0,
    int64_t* input_pointer, size_t input_dim_c0, size_t input_dim_c1,
    size_t input_dim_c2, size_t number_of_spikes, size_t dim_x, size_t dim_y,
    float forgetting_offset, float forgetting_offset_local, size_t pattern_id,
    bool approximation_multiplication_enable, uint64_t number_of_frac_bits,
    bool approximation_enable, uint64_t number_of_trunc_bits,
    uint32_t ap_mask) {
  float* h_ptr;
  float* epsilon_xy_ptr;
  int64_t* input_ptr;

  size_t counter_x;
  size_t counter_y;

  for (counter_x = 0; counter_x < dim_x; counter_x++) {
    for (counter_y = 0; counter_y < dim_y; counter_y++) {
      epsilon_xy_ptr =
          epsilon_xy_pointer + counter_x * epsilon_xy_dim_c1 + counter_y;

      h_ptr =
          h_pointer + pattern_id * h_dim_c0 + counter_x * h_dim_c2 + counter_y;

      input_ptr = input_pointer + pattern_id * input_dim_c0 +
                  counter_x * input_dim_c2 + counter_y;

      if (approximation_multiplication_enable == false) {
        update_one_ip(h_init_ptr, h_ptr, h_dim_c1, h_dim, weights_pointer,
                      weights_dim_c0, input_ptr, input_dim_c1, epsilon_xy_ptr,
                      epsilon_xy_dim_c0, epsilon_t_pointer, number_of_spikes,
                      forgetting_offset, forgetting_offset_local);
      } else {
        update_one_ip_approx(
            h_init_ptr, h_ptr, h_dim_c1, h_dim, weights_pointer, weights_dim_c0,
            input_ptr, input_dim_c1, epsilon_xy_ptr, epsilon_xy_dim_c0,
            epsilon_t_pointer, number_of_spikes, forgetting_offset,
            forgetting_offset_local, approximation_multiplication_enable,
            number_of_frac_bits, approximation_enable, number_of_trunc_bits,
            ap_mask);
      }
    }
  }

  return true;
};

void HDynamicCNNManyIP::update_one_ip_approx(
    float* h_init_ptr, float* h_pointer, size_t h_dim_c1, size_t h_dim,
    float* weights_pointer, size_t weights_dim_c0, int64_t* input_pointer,
    size_t input_dim_c1, float* epsilon_xy_pointer, size_t epsilon_xy_dim_c0,
    float* epsilon_t_pointer, size_t number_of_spikes, float forgetting_offset,
    float forgetting_offset_local, bool approximation_multiplication_enable,
    uint64_t number_of_frac_bits, bool approximation_enable,
    uint64_t number_of_trunc_bits, uint32_t ap_mask) {
  float* h_temp = new float[h_dim];
  float* h_subsegment = new float[h_dim];

  memcpy(h_subsegment, h_init_ptr, sizeof(float) * h_dim);

  size_t counter_spike;
  size_t counter;

  float h_temp_sum;
  float temp_value;

  float epsilon_subsegment;
  float epsilon_scale = 1.0;

  int64_t* spike;
  float* w_ptr;

  // ---------------
  // Approx...

  uint64_t pattern_size = h_dim;

  std::vector<float> ap_h_vector;
  ap_h_vector.resize(pattern_size);
  float* ap_h_ptr = ap_h_vector.data();

  std::vector<uint32_t> ap_x_vector;
  ap_x_vector.resize(pattern_size);
  uint32_t* ap_x_ptr = ap_x_vector.data();

  std::vector<uint32_t> ap_y_vector;
  ap_y_vector.resize(pattern_size);
  uint32_t* ap_y_ptr = ap_y_vector.data();

  std::vector<uint32_t> ap_x_exponent_vector;
  ap_x_exponent_vector.resize(pattern_size);
  uint32_t* ap_x_exponent_ptr = ap_x_exponent_vector.data();

  std::vector<uint32_t> ap_y_exponent_vector;
  ap_y_exponent_vector.resize(pattern_size);
  uint32_t* ap_y_exponent_ptr = ap_y_exponent_vector.data();

  std::vector<uint32_t> ap_h_exponent_vector;
  ap_h_exponent_vector.resize(pattern_size);
  uint32_t* ap_h_exponent_ptr = ap_h_exponent_vector.data();

  std::vector<uint64_t> ap_res_vector;
  ap_res_vector.resize(pattern_size);
  uint64_t* ap_res_ptr = ap_res_vector.data();

  std::vector<uint32_t> sign_temp_vector;
  sign_temp_vector.resize(pattern_size);
  uint32_t* sign_temp_ptr = sign_temp_vector.data();

  // --------------

  for (counter_spike = 0; counter_spike < number_of_spikes; counter_spike++) {
    if (epsilon_scale > 1E10) {
      temp_value = 1.0 / epsilon_scale;

#pragma omp simd
      for (counter = 0; counter < h_dim; counter++) {
        h_subsegment[counter] *= temp_value;
      }

      epsilon_scale = 1.0;
    }

    spike = input_pointer + counter_spike * input_dim_c1;

    if (*spike >= 0) {
      epsilon_subsegment = epsilon_xy_pointer[*spike * epsilon_xy_dim_c0] *
                           epsilon_t_pointer[counter_spike];

      w_ptr = weights_pointer + *spike * weights_dim_c0;

      memcpy(h_temp, h_subsegment, sizeof(float) * h_dim);

      approximation_multiplication_function(
          ap_h_ptr, w_ptr, pattern_size, number_of_trunc_bits,
          number_of_frac_bits, ap_x_ptr, ap_y_ptr, ap_x_exponent_ptr,
          ap_y_exponent_ptr, ap_h_exponent_ptr, ap_mask, ap_res_ptr,
          sign_temp_ptr, approximation_enable);
      // --------------------------

      h_temp_sum = 0.0;
#pragma omp simd reduction(+ : h_temp_sum)
      for (counter = 0; counter < h_dim; counter++) {
        h_temp_sum += h_temp[counter];
      }

      if (h_temp_sum > 1E-10) {
        temp_value = epsilon_scale * epsilon_subsegment / h_temp_sum;

#pragma omp simd
        for (counter = 0; counter < h_dim; counter++) {
          h_temp[counter] *= temp_value;
        }

#pragma omp simd
        for (counter = 0; counter < h_dim; counter++) {
          h_subsegment[counter] += h_temp[counter];
        }

        if (forgetting_offset_local > 0.0) {
          temp_value =
              epsilon_scale * epsilon_subsegment * forgetting_offset_local;

#pragma omp simd
          for (counter = 0; counter < h_dim; counter++) {
            h_subsegment[counter] += temp_value;
          }

          epsilon_scale *= 1.0 + epsilon_subsegment * (1.0 + forgetting_offset);
        } else {
          epsilon_scale *= 1.0 + epsilon_subsegment * 1.0;
        }
      }
    }
  }

  temp_value = 1.0 / epsilon_scale;
#pragma omp simd
  for (counter = 0; counter < h_dim; counter++) {
    h_pointer[counter * h_dim_c1] = h_subsegment[counter] * temp_value;
  }

  delete[] h_temp;
  delete[] h_subsegment;

  return;
};

void HDynamicCNNManyIP::update_one_ip(
    float* h_init_ptr, float* h_pointer, size_t h_dim_c1, size_t h_dim,
    float* weights_pointer, size_t weights_dim_c0, int64_t* input_pointer,
    size_t input_dim_c1, float* epsilon_xy_pointer, size_t epsilon_xy_dim_c0,
    float* epsilon_t_pointer, size_t number_of_spikes, float forgetting_offset,
    float forgetting_offset_local) {
  float* h_temp = new float[h_dim];
  float* h_subsegment = new float[h_dim];

  memcpy(h_subsegment, h_init_ptr, sizeof(float) * h_dim);

  size_t counter_spike;
  size_t counter;

  float h_temp_sum;
  float temp_value;

  float epsilon_subsegment;
  float epsilon_scale = 1.0;

  int64_t* spike;
  float* w_ptr;

  // --------------

  for (counter_spike = 0; counter_spike < number_of_spikes; counter_spike++) {
    if (epsilon_scale > 1E10) {
      temp_value = 1.0 / epsilon_scale;

#pragma omp simd
      for (counter = 0; counter < h_dim; counter++) {
        h_subsegment[counter] *= temp_value;
      }

      epsilon_scale = 1.0;
    }

    spike = input_pointer + counter_spike * input_dim_c1;

    if (*spike >= 0) {
      epsilon_subsegment = epsilon_xy_pointer[*spike * epsilon_xy_dim_c0] *
                           epsilon_t_pointer[counter_spike];

      w_ptr = weights_pointer + *spike * weights_dim_c0;

      memcpy(h_temp, h_subsegment, sizeof(float) * h_dim);

#pragma omp simd
      for (counter = 0; counter < h_dim; counter++) {
        h_temp[counter] *= w_ptr[counter];
      }

      h_temp_sum = 0.0;
#pragma omp simd reduction(+ : h_temp_sum)
      for (counter = 0; counter < h_dim; counter++) {
        h_temp_sum += h_temp[counter];
      }

      if (h_temp_sum > 1E-10) {
        temp_value = epsilon_scale * epsilon_subsegment / h_temp_sum;

#pragma omp simd
        for (counter = 0; counter < h_dim; counter++) {
          h_temp[counter] *= temp_value;
        }

#pragma omp simd
        for (counter = 0; counter < h_dim; counter++) {
          h_subsegment[counter] += h_temp[counter];
        }

        if (forgetting_offset_local > 0.0) {
          temp_value =
              epsilon_scale * epsilon_subsegment * forgetting_offset_local;

#pragma omp simd
          for (counter = 0; counter < h_dim; counter++) {
            h_subsegment[counter] += temp_value;
          }

          epsilon_scale *= 1.0 + epsilon_subsegment * (1.0 + forgetting_offset);
        } else {
          epsilon_scale *= 1.0 + epsilon_subsegment * 1.0;
        }
      }
    }
  }

  temp_value = 1.0 / epsilon_scale;
#pragma omp simd
  for (counter = 0; counter < h_dim; counter++) {
    h_pointer[counter * h_dim_c1] = h_subsegment[counter] * temp_value;
  }

  delete[] h_temp;
  delete[] h_subsegment;

  return;
};

// ------------------------------------------------

void HDynamicCNNManyIP::gpu_occupancy_measure(size_t dim_x, size_t dim_y,
                                              size_t number_of_pattern,
                                              size_t h_dim) {
  grid_and_thread_calculated = false;
  assert((dim_x < 65535));
  assert((dim_y < 65535));

  grid_and_thread_settings.resize(14);

  occupancy_kernel_phxy_plus_phxy(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PHXY], display_debug);

  occupancy_kernel_pxy_plus_v(dim_x, dim_y, number_of_pattern, h_dim,
                              grid_and_thread_settings[ID_KERNEL_PXY_PLUS_V],
                              display_debug);

  occupancy_kernel_pxy_times_v(dim_x, dim_y, number_of_pattern, h_dim,
                               grid_and_thread_settings[ID_KERNEL_PXY_TIMES_V],
                               display_debug);

  occupancy_kernel_phxy_fill_with_h(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_H], display_debug);

  occupancy_kernel_phxy_plus_pxy(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PXY], display_debug);

  occupancy_kernel_pxy_reciprocal(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PXY_RECIPROCAL], display_debug);

  occupancy_kernel_phxy_fill_with_spike_selected_w(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_SPIKE_SELECTED_W],
      display_debug);

  occupancy_kernel_phxy_times_phxy_equals_phxy(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PHXY_EQUALS_PHXY],
      display_debug);

  occupancy_kernel_pxy_set_to_v(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V], display_debug);

  occupancy_kernel_phxy_one_over_sum_into_pxy(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PHXY_ONE_OVER_SUM_INTO_PXY],
      display_debug);

  occupancy_kernel_phxy_times_pxy(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY], display_debug);

  occupancy_kernel_pxy_time_pxy(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY], display_debug);

  occupancy_kernel_approximation_pure_multiplication(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_APPROXIMATION_MULTIPLICATION],
      display_debug);

  occupancy_kernel_pxy_times_spike_selected_sxy(
      dim_x, dim_y, number_of_pattern, h_dim,
      grid_and_thread_settings[ID_KERNEL_PXY_TIMES_SPIKE_SELECTED_SXY],
      display_debug);

  grid_and_thread_calculated = true;
  return;
};

void HDynamicCNNManyIP::gpu_occupancy_export(
    size_t dim_x, size_t dim_y, size_t number_of_pattern, size_t h_dim,
    int64_t setting_memory_addr, size_t setting_dim_0, size_t setting_dim_1) {
  int64_t* setting_memory = (int64_t*)setting_memory_addr;

  assert((setting_memory != nullptr));
  assert((setting_dim_1 == H_DYNAMIC_NUMBER_OF_KERNELS_PARAMETERS));

  gpu_occupancy_measure(dim_x, dim_y, number_of_pattern, h_dim);
  assert((grid_and_thread_calculated == true));

  assert((setting_dim_0 == grid_and_thread_settings.size()));

  for (size_t counter_0 = 0; counter_0 < setting_dim_0; counter_0++) {
    for (size_t counter_1 = 0; counter_1 < setting_dim_1; counter_1++) {
      setting_memory[counter_0 * setting_dim_1 + counter_1] =
          grid_and_thread_settings[counter_0][counter_1];
    }
  }
};

void HDynamicCNNManyIP::gpu_occupancy_import(int64_t setting_memory_addr,
                                             size_t setting_dim_0,
                                             size_t setting_dim_1) {
  grid_and_thread_calculated = false;

  int64_t* setting_memory = (int64_t*)setting_memory_addr;

  assert((setting_memory != nullptr));
  assert((setting_dim_1 == H_DYNAMIC_NUMBER_OF_KERNELS_PARAMETERS));
  assert((setting_dim_0 == H_DYNAMIC_NUMBER_OF_KERNELS));

  grid_and_thread_settings.resize(H_DYNAMIC_NUMBER_OF_KERNELS);

  for (size_t counter_0 = 0; counter_0 < setting_dim_0; counter_0++) {
    grid_and_thread_settings[counter_0].resize(
        H_DYNAMIC_NUMBER_OF_KERNELS_PARAMETERS);

    for (size_t counter_1 = 0; counter_1 < setting_dim_1; counter_1++) {
      grid_and_thread_settings[counter_0][counter_1] =
          setting_memory[counter_0 * setting_dim_1 + counter_1];
    }
  }

  grid_and_thread_calculated = true;
};

bool HDynamicCNNManyIP::gpu_update(
    float* h_init_ptr, float* h_pointer, size_t h_dim_c0, size_t h_dim_c1,
    size_t h_dim_c2, size_t h_dim, float* epsilon_xy_pointer,
    size_t epsilon_xy_dim_c0, size_t epsilon_xy_dim_c1,
    float* epsilon_t_pointer, float* weights_pointer, size_t weights_dim_c0,
    int64_t* input_pointer, size_t input_dim_c0, size_t input_dim_c1,
    size_t input_dim_c2, size_t number_of_spikes, size_t dim_x, size_t dim_y,
    float forgetting_offset, float forgetting_offset_local,
    size_t number_of_pattern, size_t gpu_tuning_factor,
    bool approximation_multiplication_enable, uint64_t number_of_frac_bits,
    bool approximation_enable, uint64_t number_of_trunc_bits,
    uint32_t ap_mask) {
  if (grid_and_thread_calculated == false) {
    gpu_occupancy_measure(dim_x, dim_y, number_of_pattern, h_dim);
  }
  assert((grid_and_thread_calculated == true));

  hipError_t status;

  size_t h_sum_dim_c0 = dim_x * dim_y;
  size_t h_sum_dim_c1 = dim_y;

  size_t phxy_block_dim_c0 = h_dim * dim_x * dim_y;
  size_t phxy_block_dim_c1 = dim_x * dim_y;
  size_t phxy_block_dim_c2 = dim_y;

  size_t pxy_block_dim_c0 = dim_x * dim_y;
  size_t pxy_block_dim_c1 = dim_y;

  float* w_memory = nullptr;
  status = hipMalloc((void**)&w_memory, number_of_pattern * h_dim * dim_x *
                                             dim_y * sizeof(float));
  assert((status == hipSuccess));

  float* h_temp_memory = nullptr;
  status =
      hipMalloc((void**)&h_temp_memory,
                 number_of_pattern * h_dim * dim_x * dim_y * sizeof(float));
  assert((status == hipSuccess));

  float* h_sum_memory = nullptr;
  status = hipMalloc((void**)&h_sum_memory,
                      number_of_pattern * dim_x * dim_y * sizeof(float));
  assert((status == hipSuccess));

  float* epsilon_subsegment_memory = nullptr;
  status = hipMalloc((void**)&epsilon_subsegment_memory,
                      number_of_pattern * dim_x * dim_y * sizeof(float));
  assert((status == hipSuccess));

  float* epsilon_scale_memory = nullptr;
  status = hipMalloc((void**)&epsilon_scale_memory,
                      number_of_pattern * dim_x * dim_y * sizeof(float));
  assert((status == hipSuccess));

  float* forget_memory = nullptr;
  status = hipMalloc((void**)&forget_memory,
                      number_of_pattern * dim_x * dim_y * sizeof(float));
  assert((status == hipSuccess));

  // ---

  // Initialize h
  kernel_phxy_fill_with_h<<<
      dim3(grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_H][0],
           grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_H][1],
           grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_H][2]),
      dim3(grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_H][3],
           grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_H][4],
           grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_H][5])>>>(
      h_init_ptr, h_pointer, h_dim_c0, h_dim_c1, h_dim_c2, h_dim,
      phxy_block_dim_c0, phxy_block_dim_c1, phxy_block_dim_c2,
      grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_H][6]);
  status = hipDeviceSynchronize();
  assert((status == hipSuccess));

  // Set epsilon memory scale to 1.0
  kernel_pxy_set_to_v<<<
      dim3(grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][0],
           grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][1],
           grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][2]),
      dim3(grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][3],
           grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][4],
           grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][5])>>>(
      epsilon_scale_memory, 1.0,
      grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][6]);
  status = hipDeviceSynchronize();
  assert((status == hipSuccess));

  for (size_t counter_spike = 0; counter_spike < number_of_spikes;
       counter_spike++) {
    // Get epsilon_t from gpu memory
    float epsilon_t;
    status = hipMemcpy(&epsilon_t, &epsilon_t_pointer[counter_spike],
                        sizeof(float), hipMemcpyDeviceToHost);
    assert((status == hipSuccess));
    // Set epsilon memory subsegment to epsilon(t)
    kernel_pxy_set_to_v<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][0],
             grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][1],
             grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][3],
             grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][4],
             grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][5])>>>(
        epsilon_subsegment_memory, epsilon_t,
        grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // Set epsilon memory subsegment to forgetting_offset_local
    kernel_pxy_set_to_v<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][0],
             grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][1],
             grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][3],
             grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][4],
             grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][5])>>>(
        forget_memory, forgetting_offset_local,
        grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    //     if (*spike >= 0) {
    //       epsilon_subsegment = *epsilon_xy_pointer[*spike *
    //       epsilon_xy_dim_c0]
    kernel_pxy_times_spike_selected_sxy<<<
        dim3(
            grid_and_thread_settings[ID_KERNEL_PXY_TIMES_SPIKE_SELECTED_SXY][0],
            grid_and_thread_settings[ID_KERNEL_PXY_TIMES_SPIKE_SELECTED_SXY][1],
            grid_and_thread_settings[ID_KERNEL_PXY_TIMES_SPIKE_SELECTED_SXY]
                                    [2]),
        dim3(
            grid_and_thread_settings[ID_KERNEL_PXY_TIMES_SPIKE_SELECTED_SXY][3],
            grid_and_thread_settings[ID_KERNEL_PXY_TIMES_SPIKE_SELECTED_SXY][4],
            grid_and_thread_settings[ID_KERNEL_PXY_TIMES_SPIKE_SELECTED_SXY]
                                    [5])>>>(
        epsilon_subsegment_memory, epsilon_xy_pointer, input_pointer,
        counter_spike, input_dim_c0, input_dim_c1, input_dim_c2,
        epsilon_xy_dim_c0, epsilon_xy_dim_c1, epsilon_xy_dim_c0,
        epsilon_xy_dim_c1, pxy_block_dim_c0, pxy_block_dim_c1,
        grid_and_thread_settings[ID_KERNEL_PXY_TIMES_SPIKE_SELECTED_SXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // Get the weight vectors according the spikes
    kernel_phxy_fill_with_spike_selected_w<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_SPIKE_SELECTED_W]
                                     [0],
             grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_SPIKE_SELECTED_W]
                                     [1],
             grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_SPIKE_SELECTED_W]
                                     [2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_SPIKE_SELECTED_W]
                                     [3],
             grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_SPIKE_SELECTED_W]
                                     [4],
             grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_SPIKE_SELECTED_W]
                                     [5])>>>(
        w_memory, weights_pointer, input_pointer, counter_spike, weights_dim_c0,
        input_dim_c0, input_dim_c1, input_dim_c2, h_dim_c0, h_dim_c1, h_dim_c2,
        h_dim, phxy_block_dim_c0, phxy_block_dim_c1, phxy_block_dim_c2,
        grid_and_thread_settings[ID_KERNEL_PHXY_FILL_WITH_SPIKE_SELECTED_W][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // h_temp = h * w
    if (approximation_multiplication_enable == false) {
      kernel_phxy_times_phxy_equals_phxy<<<
          dim3(grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PHXY_EQUALS_PHXY]
                                       [0],
               grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PHXY_EQUALS_PHXY]
                                       [1],
               grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PHXY_EQUALS_PHXY]
                                       [2]),
          dim3(grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PHXY_EQUALS_PHXY]
                                       [3],
               grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PHXY_EQUALS_PHXY]
                                       [4],
               grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PHXY_EQUALS_PHXY]
                                       [5])>>>(
          h_pointer, w_memory, h_temp_memory,
          grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PHXY_EQUALS_PHXY][6]);

    } else {
      kernel_approximation_pure_multiplication<<<
          dim3(grid_and_thread_settings[ID_KERNEL_APPROXIMATION_MULTIPLICATION]
                                       [0],
               grid_and_thread_settings[ID_KERNEL_APPROXIMATION_MULTIPLICATION]
                                       [1],
               grid_and_thread_settings[ID_KERNEL_APPROXIMATION_MULTIPLICATION]
                                       [2]),
          dim3(grid_and_thread_settings[ID_KERNEL_APPROXIMATION_MULTIPLICATION]
                                       [3],
               grid_and_thread_settings[ID_KERNEL_APPROXIMATION_MULTIPLICATION]
                                       [4],
               grid_and_thread_settings[ID_KERNEL_APPROXIMATION_MULTIPLICATION]
                                       [5])>>>(
          h_pointer, w_memory, h_temp_memory, number_of_frac_bits,
          approximation_enable, number_of_trunc_bits, ap_mask,
          grid_and_thread_settings[ID_KERNEL_APPROXIMATION_MULTIPLICATION][6]);
    }

    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // 1 / sum h_temp
    kernel_phxy_one_over_sum_into_pxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_ONE_OVER_SUM_INTO_PXY][0],
             grid_and_thread_settings[ID_KERNEL_PHXY_ONE_OVER_SUM_INTO_PXY][1],
             grid_and_thread_settings[ID_KERNEL_PHXY_ONE_OVER_SUM_INTO_PXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_ONE_OVER_SUM_INTO_PXY][3],
             grid_and_thread_settings[ID_KERNEL_PHXY_ONE_OVER_SUM_INTO_PXY][4],
             grid_and_thread_settings[ID_KERNEL_PHXY_ONE_OVER_SUM_INTO_PXY]
                                     [5])>>>(
        h_temp_memory, h_sum_memory, h_dim_c0, h_dim_c1, h_dim_c2, h_dim,
        h_sum_dim_c0, h_sum_dim_c1, pxy_block_dim_c0, pxy_block_dim_c1,
        grid_and_thread_settings[ID_KERNEL_PHXY_ONE_OVER_SUM_INTO_PXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // epsilon_scale / sum h_temp
    kernel_pxy_time_pxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][0],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][1],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][3],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][4],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][5])>>>(
        h_sum_memory, epsilon_scale_memory,
        grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // epsilon_subsegment * epsilon_scale / sum h_temp
    kernel_pxy_time_pxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][0],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][1],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][3],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][4],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][5])>>>(
        h_sum_memory, epsilon_subsegment_memory,
        grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // epsilon_scale * forget_memory which contains forgetting_offset_local
    kernel_pxy_time_pxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][0],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][1],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][3],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][4],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][5])>>>(
        forget_memory, epsilon_scale_memory,
        grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // delta_forget = epsilon_subsegment * epsilon_scale * forget_memory
    kernel_pxy_time_pxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][0],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][1],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][3],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][4],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][5])>>>(
        forget_memory, epsilon_subsegment_memory,
        grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // delta_h = h_temp_memory * epsilon_subsegment * epsilon_scale / sum h
    kernel_phxy_times_pxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][0],
             grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][1],
             grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][3],
             grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][4],
             grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][5])>>>(
        h_temp_memory, h_sum_memory, h_dim_c0, h_dim_c1, h_dim_c2, h_dim,
        h_sum_dim_c0, h_sum_dim_c1, phxy_block_dim_c0, phxy_block_dim_c1,
        phxy_block_dim_c2,
        grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // h + delta_h
    kernel_phxy_plus_phxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PHXY][0],
             grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PHXY][1],
             grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PHXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PHXY][3],
             grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PHXY][4],
             grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PHXY][5])>>>(
        h_pointer, h_temp_memory,
        grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PHXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // h + delta_h + delta_forget
    kernel_phxy_plus_pxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PXY][0],
             grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PXY][1],
             grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PXY][3],
             grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PXY][4],
             grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PXY][5])>>>(
        h_pointer, forget_memory, h_dim_c0, h_dim_c1, h_dim_c2, h_dim,
        h_sum_dim_c0, h_sum_dim_c1, phxy_block_dim_c0, phxy_block_dim_c1,
        phxy_block_dim_c2,
        grid_and_thread_settings[ID_KERNEL_PHXY_PLUS_PXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    kernel_pxy_times_v<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIMES_V][0],
             grid_and_thread_settings[ID_KERNEL_PXY_TIMES_V][1],
             grid_and_thread_settings[ID_KERNEL_PXY_TIMES_V][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIMES_V][3],
             grid_and_thread_settings[ID_KERNEL_PXY_TIMES_V][4],
             grid_and_thread_settings[ID_KERNEL_PXY_TIMES_V][5])>>>(
        epsilon_subsegment_memory, (1.0 + forgetting_offset),
        grid_and_thread_settings[ID_KERNEL_PXY_TIMES_V][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    kernel_pxy_plus_v<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_PLUS_V][0],
             grid_and_thread_settings[ID_KERNEL_PXY_PLUS_V][1],
             grid_and_thread_settings[ID_KERNEL_PXY_PLUS_V][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_PLUS_V][3],
             grid_and_thread_settings[ID_KERNEL_PXY_PLUS_V][4],
             grid_and_thread_settings[ID_KERNEL_PXY_PLUS_V][5])>>>(
        epsilon_subsegment_memory, 1.0,
        grid_and_thread_settings[ID_KERNEL_PXY_PLUS_V][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    // epsilon_scale * epsilon_subsegment
    kernel_pxy_time_pxy<<<
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][0],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][1],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][2]),
        dim3(grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][3],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][4],
             grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][5])>>>(
        epsilon_scale_memory, epsilon_subsegment_memory,
        grid_and_thread_settings[ID_KERNEL_PXY_TIME_PXY][6]);
    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    if (((counter_spike > 0) && (counter_spike % 1000 == 0)) ||
        (counter_spike + 1 == number_of_spikes)) {
      kernel_pxy_reciprocal<<<
          dim3(grid_and_thread_settings[ID_KERNEL_PXY_RECIPROCAL][0],
               grid_and_thread_settings[ID_KERNEL_PXY_RECIPROCAL][1],
               grid_and_thread_settings[ID_KERNEL_PXY_RECIPROCAL][2]),
          dim3(grid_and_thread_settings[ID_KERNEL_PXY_RECIPROCAL][3],
               grid_and_thread_settings[ID_KERNEL_PXY_RECIPROCAL][4],
               grid_and_thread_settings[ID_KERNEL_PXY_RECIPROCAL][5])>>>(
          epsilon_scale_memory,
          grid_and_thread_settings[ID_KERNEL_PXY_RECIPROCAL][6]);
      status = hipDeviceSynchronize();
      assert((status == hipSuccess));

      kernel_phxy_times_pxy<<<
          dim3(grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][0],
               grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][1],
               grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][2]),
          dim3(grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][3],
               grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][4],
               grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][5])>>>(
          h_pointer, epsilon_scale_memory, h_dim_c0, h_dim_c1, h_dim_c2, h_dim,
          h_sum_dim_c0, h_sum_dim_c1, phxy_block_dim_c0, phxy_block_dim_c1,
          phxy_block_dim_c2,
          grid_and_thread_settings[ID_KERNEL_PHXY_TIMES_PXY][6]);
      status = hipDeviceSynchronize();
      assert((status == hipSuccess));

      // Set epsilon memory scale to 1.0
      kernel_pxy_set_to_v<<<
          dim3(grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][0],
               grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][1],
               grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][2]),
          dim3(grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][3],
               grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][4],
               grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][5])>>>(
          epsilon_scale_memory, 1.0,
          grid_and_thread_settings[ID_KERNEL_PXY_SET_TO_V][6]);
      status = hipDeviceSynchronize();
      assert((status == hipSuccess));
    }
  }

  // ------------

  status = hipFree(w_memory);
  assert((status == hipSuccess));

  status = hipFree(h_temp_memory);
  assert((status == hipSuccess));

  status = hipFree(h_sum_memory);
  assert((status == hipSuccess));

  status = hipFree(epsilon_subsegment_memory);
  assert((status == hipSuccess));

  status = hipFree(epsilon_scale_memory);
  assert((status == hipSuccess));

  status = hipFree(forget_memory);
  assert((status == hipSuccess));

  return true;
};