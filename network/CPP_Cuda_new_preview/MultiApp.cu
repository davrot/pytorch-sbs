#include "hip/hip_runtime.h"
#include <omp.h>
#include <stdio.h>
#include <string.h>

#include <algorithm>
#include <cassert>
#include <cmath>
#include <iostream>
#include <vector>

#include "MultiApp.h"
#include "approximation_multiplication_function.h"
#include "kernel_approximation_multiplication.h"

MultiApp::MultiApp(){

};

MultiApp::~MultiApp(){

};

bool MultiApp::update(float* np_input_pointer, float* np_weight_pointer,
                      float* np_output_pointer, int64_t pattern_dim,
                      int64_t feature_dim, int64_t x_dim, int64_t y_dim,
                      int64_t input_channel_dim, int64_t id_pattern,
                      bool approximation_enable, int64_t number_of_trunc_bits,
                      int64_t number_of_frac_bits) {
  assert((id_pattern >= 0));
  assert((id_pattern < pattern_dim));

  float* np_input_pointer_pattern;
  float* np_output_pointer_pattern;

  float* input_ptr;
  float* output_ptr;
  float* w_ptr;

  uint64_t pattern_size = input_channel_dim;

  std::vector<float> ap_h_vector;
  ap_h_vector.resize(pattern_size);
  float* ap_h_ptr = ap_h_vector.data();

  std::vector<uint32_t> ap_x_vector;
  ap_x_vector.resize(pattern_size);
  uint32_t* ap_x_ptr = ap_x_vector.data();

  std::vector<uint32_t> ap_y_vector;
  ap_y_vector.resize(pattern_size);
  uint32_t* ap_y_ptr = ap_y_vector.data();

  std::vector<uint32_t> ap_x_exponent_vector;
  ap_x_exponent_vector.resize(pattern_size);
  uint32_t* ap_x_exponent_ptr = ap_x_exponent_vector.data();

  std::vector<uint32_t> ap_y_exponent_vector;
  ap_y_exponent_vector.resize(pattern_size);
  uint32_t* ap_y_exponent_ptr = ap_y_exponent_vector.data();

  std::vector<uint32_t> ap_h_exponent_vector;
  ap_h_exponent_vector.resize(pattern_size);
  uint32_t* ap_h_exponent_ptr = ap_h_exponent_vector.data();

  std::vector<uint64_t> ap_res_vector;
  ap_res_vector.resize(pattern_size);
  uint64_t* ap_res_ptr = ap_res_vector.data();

  uint32_t ap_mask = static_cast<uint64_t>(pow(2, number_of_trunc_bits)) - 1;

  std::vector<uint32_t> sign_temp_vector;
  sign_temp_vector.resize(pattern_size);
  uint32_t* sign_temp_ptr = sign_temp_vector.data();

  uint64_t input_pattern_size = input_channel_dim * x_dim * y_dim;
  uint64_t output_pattern_size = feature_dim * x_dim * y_dim;

  np_input_pointer_pattern = np_input_pointer + id_pattern * input_pattern_size;
  np_output_pointer_pattern =
      np_output_pointer + id_pattern * output_pattern_size;

  uint64_t counter;

  uint64_t counter_x;
  uint64_t counter_y;
  uint64_t counter_feature;
  uint64_t pos_xy;
  uint64_t pos_xy_if;

  float temp_sum;

  uint64_t pattern_c_2 = x_dim * y_dim;

  for (counter_x = 0; counter_x < x_dim; counter_x++) {
    for (counter_y = 0; counter_y < y_dim; counter_y++) {
      pos_xy = counter_y + counter_x * y_dim;
      for (counter_feature = 0; counter_feature < feature_dim;
           counter_feature++) {
        pos_xy_if = counter_feature * pattern_c_2 + pos_xy;

        input_ptr = np_input_pointer_pattern + pos_xy;
        output_ptr = np_output_pointer_pattern + pos_xy_if;
        w_ptr = np_weight_pointer + counter_feature * input_channel_dim;

#pragma omp simd
        for (counter = 0; counter < pattern_size; counter++) {
          ap_h_ptr[counter] = input_ptr[counter * pattern_c_2];
        }

        approximation_multiplication_function(
            ap_h_ptr, w_ptr, pattern_size, number_of_trunc_bits,
            number_of_frac_bits, ap_x_ptr, ap_y_ptr, ap_x_exponent_ptr,
            ap_y_exponent_ptr, ap_h_exponent_ptr, ap_mask, ap_res_ptr,
            sign_temp_ptr, approximation_enable);

        temp_sum = 0.0;
#pragma omp simd reduction(+ : temp_sum)
        for (counter = 0; counter < pattern_size; counter++) {
          temp_sum += ap_h_ptr[counter];
        }

        output_ptr[0] = temp_sum;
      }
    }
  }

  return true;
};

bool MultiApp::update_entrypoint(
    int64_t np_input_pointer_addr, int64_t np_weight_pointer_addr,
    int64_t np_output_pointer_addr, int64_t pattern_dim, int64_t feature_dim,
    int64_t x_dim, int64_t y_dim, int64_t input_channel_dim,
    int64_t number_of_processes, bool approximation_enable,
    int64_t number_of_trunc_bits, int64_t number_of_frac) {
  int64_t number_of_pattern = pattern_dim;
  int64_t pattern_id;

  float* np_input_pointer = (float*)np_input_pointer_addr;
  float* np_weight_pointer = (float*)np_weight_pointer_addr;
  float* np_output_pointer = (float*)np_output_pointer_addr;

  assert((np_input_pointer != nullptr));
  assert((np_output_pointer != nullptr));
  assert((np_weight_pointer != nullptr));

  assert((pattern_dim > 0));
  assert((feature_dim > 0));
  assert((x_dim > 0));
  assert((y_dim > 0));
  assert((input_channel_dim > 0));

  if (number_of_processes > 0) {
    omp_set_num_threads(number_of_processes);
    // For debugging: Only one thread
    // omp_set_num_threads(1);

#pragma omp parallel for
    for (pattern_id = 0; pattern_id < number_of_pattern; pattern_id++) {
      update(np_input_pointer, np_weight_pointer, np_output_pointer,
             pattern_dim, feature_dim, x_dim, y_dim, input_channel_dim,
             pattern_id, approximation_enable, number_of_trunc_bits,
             number_of_frac);
    }
  } else {
    update_gpu(np_input_pointer, np_weight_pointer, np_output_pointer,
               pattern_dim, feature_dim, x_dim, y_dim, input_channel_dim,
               approximation_enable, number_of_trunc_bits, number_of_frac);
  }
  return true;
};

void MultiApp::gpu_occupancy_measure(size_t dim_x, size_t dim_y,
                                     size_t number_of_pattern, size_t h_dim) {
  grid_and_thread_calculated = false;
  assert((dim_x < 65535));
  assert((dim_y < 65535));

  grid_and_thread_settings.resize(1);

  occupancy_kernel_approximation_multiplication(
      dim_x, dim_y, number_of_pattern, h_dim, grid_and_thread_settings[0],
      display_debug);

  grid_and_thread_calculated = true;
  return;
};

void MultiApp::gpu_occupancy_export(size_t dim_x, size_t dim_y,
                                    size_t number_of_pattern, size_t h_dim,
                                    int64_t setting_memory_addr,
                                    size_t setting_dim_0,
                                    size_t setting_dim_1) {
  int64_t* setting_memory = (int64_t*)setting_memory_addr;

  assert((setting_memory != nullptr));
  assert((setting_dim_1 == APPROXI_MULTI_NUMBER_OF_KERNELS_PARAMETERS));

  gpu_occupancy_measure(dim_x, dim_y, number_of_pattern, h_dim);
  assert((grid_and_thread_calculated == true));

  assert((setting_dim_0 == grid_and_thread_settings.size()));

  for (size_t counter_0 = 0; counter_0 < setting_dim_0; counter_0++) {
    for (size_t counter_1 = 0; counter_1 < setting_dim_1; counter_1++) {
      setting_memory[counter_0 * setting_dim_1 + counter_1] =
          grid_and_thread_settings[counter_0][counter_1];
    }
  }
};

void MultiApp::gpu_occupancy_import(int64_t setting_memory_addr,
                                    size_t setting_dim_0,
                                    size_t setting_dim_1) {
  grid_and_thread_calculated = false;

  int64_t* setting_memory = (int64_t*)setting_memory_addr;

  assert((setting_memory != nullptr));
  assert((setting_dim_1 == APPROXI_MULTI_NUMBER_OF_KERNELS_PARAMETERS));
  assert((setting_dim_0 == APPROXI_MULTI_NUMBER_OF_KERNELS));

  grid_and_thread_settings.resize(APPROXI_MULTI_NUMBER_OF_KERNELS);

  for (size_t counter_0 = 0; counter_0 < setting_dim_0; counter_0++) {
    grid_and_thread_settings[counter_0].resize(
        APPROXI_MULTI_NUMBER_OF_KERNELS_PARAMETERS);

    for (size_t counter_1 = 0; counter_1 < setting_dim_1; counter_1++) {
      grid_and_thread_settings[counter_0][counter_1] =
          setting_memory[counter_0 * setting_dim_1 + counter_1];
    }
  }

  grid_and_thread_calculated = true;
};

void MultiApp::update_gpu(float* np_input_pointer, float* np_weight_pointer,
                          float* np_output_pointer, uint64_t pattern_dim,
                          uint64_t feature_dim, uint64_t x_dim, uint64_t y_dim,
                          uint64_t input_channel_dim, bool approximation_enable,
                          uint64_t number_of_trunc_bits,
                          uint64_t number_of_frac_bits) {
  if (grid_and_thread_calculated == false) {
    gpu_occupancy_measure(x_dim, y_dim, pattern_dim, feature_dim);
  }
  assert((grid_and_thread_calculated == true));

  uint32_t ap_mask = static_cast<uint64_t>(pow(2, number_of_trunc_bits)) - 1;
  // std::cout << approximation_enable << std::endl;
  // std::cout << number_of_trunc_bits << std::endl;
  // std::cout << number_of_frac_bits << std::endl;

  hipError_t status;

  size_t pfxy_block_dim_c0 = feature_dim * x_dim * y_dim;
  size_t pfxy_block_dim_c1 = x_dim * y_dim;
  size_t pfxy_block_dim_c2 = y_dim;

  kernel_approximation_multiplication<<<
      dim3(grid_and_thread_settings[0][0], grid_and_thread_settings[0][1],
           grid_and_thread_settings[0][2]),
      dim3(grid_and_thread_settings[0][3], grid_and_thread_settings[0][4],
           grid_and_thread_settings[0][5])>>>(
      np_input_pointer, np_weight_pointer, np_output_pointer, pattern_dim,
      feature_dim, x_dim, y_dim, input_channel_dim,
      grid_and_thread_settings[0][6], (x_dim * y_dim), number_of_frac_bits,
      approximation_enable, number_of_trunc_bits, ap_mask, pfxy_block_dim_c0,
      pfxy_block_dim_c1, pfxy_block_dim_c2);

  status = hipDeviceSynchronize();
  assert((status == hipSuccess));
};
