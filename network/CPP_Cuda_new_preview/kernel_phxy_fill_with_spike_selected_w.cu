#include "hip/hip_runtime.h"
#include <cassert>
#include <iostream>

#include "kernel_helper_functions.h"
#include "kernel_phxy_fill_with_spike_selected_w.h"

__global__ void kernel_phxy_fill_with_spike_selected_w(
    float* __restrict__ phxy_memory, float* __restrict__ weights_memory,
    int64_t* __restrict__ spike_memory, size_t spike_time,
    size_t weights_dim_c0, size_t spike_dim_c0, size_t spike_dim_c1,
    size_t spike_dim_c2, size_t phxy_dim_c0, size_t phxy_dim_c1,
    size_t phxy_dim_c2, size_t h_dim, size_t block_dim_c0, size_t block_dim_c1,
    size_t block_dim_c2, size_t max_idx) {
  size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < max_idx) {
    size_t pattern_id = idx / block_dim_c0;
    idx -= pattern_id * block_dim_c0;
    size_t idx_h = idx / block_dim_c1;
    idx -= idx_h * block_dim_c1;
    size_t position_x = idx / block_dim_c2;
    idx -= position_x * block_dim_c2;
    size_t position_y = idx;

    int64_t* spike = spike_memory + pattern_id * spike_dim_c0 +
                     spike_time * spike_dim_c1 + position_x * spike_dim_c2 +
                     position_y;

    if (*spike >= 0) {
      phxy_memory[pattern_id * phxy_dim_c0 + idx_h * phxy_dim_c1 +
                  position_x * phxy_dim_c2 + position_y] =
          weights_memory[*spike * weights_dim_c0 + idx_h];
    } else {
      phxy_memory[pattern_id * phxy_dim_c0 + idx_h * phxy_dim_c1 +
                  position_x * phxy_dim_c2 + position_y] = 0.0;
    }
  }
};

void occupancy_kernel_phxy_fill_with_spike_selected_w(
    size_t dim_x, size_t dim_y, size_t number_of_pattern, size_t h_dim,
    std::vector<size_t>& output, bool display_debug) {
  size_t max_threadable_tasks;
  hipError_t status;

  int min_grid_size;
  int thread_block_size;
  int grid_size;

  max_threadable_tasks = number_of_pattern * h_dim * dim_x * dim_y;

  status = hipOccupancyMaxPotentialBlockSize(
      &min_grid_size, &thread_block_size,
      (void*)kernel_phxy_fill_with_spike_selected_w, 0, max_threadable_tasks);
  assert((status == hipSuccess));

  grid_size =
      (max_threadable_tasks + thread_block_size - 1) / thread_block_size;

  output.resize(7);
  output[0] = grid_size;
  output[1] = 1;
  output[2] = 1;
  output[3] = thread_block_size;
  output[4] = 1;
  output[5] = 1;
  output[6] = max_threadable_tasks;

  if (display_debug == true) {
    std::cout << "kernel_phxy_fill_with_spike_selected_w:" << std::endl;
    kernel_debug_plot(output, display_debug);
  }
};