#include "hip/hip_runtime.h"
#include "SpikeGeneration2DManyIP.h"

#include <omp.h>
#include <stdio.h>
#include <string.h>

#include <algorithm>
#include <cassert>
#include <iostream>


SpikeGeneration2DManyIP::SpikeGeneration2DManyIP()
{

};

SpikeGeneration2DManyIP::~SpikeGeneration2DManyIP()
{

};

bool SpikeGeneration2DManyIP::spike_generation_entrypoint(
    int64_t input_pointer_addr, int64_t input_dim_0,
    int64_t input_dim_1, int64_t input_dim_2, int64_t input_dim_3,
    int64_t random_values_pointer_addr, int64_t random_values_dim_0,
    int64_t random_values_dim_1, int64_t random_values_dim_2,
    int64_t random_values_dim_3, int64_t output_pointer_addr,
    int64_t output_dim_0, int64_t output_dim_1, int64_t output_dim_2,
    int64_t output_dim_3, int64_t number_of_cpu_processes)
{

    float* input_pointer = (float*)input_pointer_addr;
    float* random_values_pointer = (float*)random_values_pointer_addr;
    int64_t* output_pointer = (int64_t*)output_pointer_addr;

    // Input
    assert((input_pointer != nullptr));
    assert((input_dim_0 > 0));
    assert((input_dim_1 > 0));
    assert((input_dim_2 > 0));
    assert((input_dim_3 > 0));

    // Random
    assert((random_values_pointer != nullptr));
    assert((random_values_dim_0 > 0));
    assert((random_values_dim_1 > 0));
    assert((random_values_dim_2 > 0));
    assert((random_values_dim_3 > 0));

    // Output
    assert((output_pointer != nullptr));
    assert((output_dim_0 > 0));
    assert((output_dim_1 > 0));
    assert((output_dim_2 > 0));
    assert((output_dim_3 > 0));

    // Input
    size_t input_dim_c0 = input_dim_1 * input_dim_2 * input_dim_3;
    size_t input_dim_c1 = input_dim_2 * input_dim_3;
    size_t input_dim_c2 = input_dim_3;

    // Random
    size_t random_values_dim_c0 =
        random_values_dim_1 * random_values_dim_2 * random_values_dim_3;
    size_t random_values_dim_c1 =
        random_values_dim_2 * random_values_dim_3;
    size_t random_values_dim_c2 = random_values_dim_3;

    // Output
    size_t output_dim_c0 =
        output_dim_1 * output_dim_2 * output_dim_3;
    size_t output_dim_c1 = output_dim_2 * output_dim_3;
    size_t output_dim_c2 = output_dim_3;

    size_t number_of_pattern = input_dim_0;
    size_t h_dim = input_dim_1;
    size_t spike_dim = output_dim_1;
    size_t x_dim = output_dim_2;
    size_t y_dim = output_dim_2;

    if (number_of_cpu_processes > 0)
    {

        omp_set_num_threads(number_of_cpu_processes);
        // DEBUG:
        // omp_set_num_threads(1);

        size_t pattern_id;

#pragma omp parallel for
        for (pattern_id = 0; pattern_id < number_of_pattern; pattern_id++)
        {
            spike_generation(
                input_pointer,
                input_dim_c0,
                input_dim_c1,
                input_dim_c2,
                random_values_pointer,
                random_values_dim_c0,
                random_values_dim_c1,
                random_values_dim_c2,
                output_pointer,
                output_dim_c0,
                output_dim_c1,
                output_dim_c2,
                x_dim,
                y_dim,
                spike_dim,
                h_dim,
                pattern_id);
        }
    }
    else
    {
        gpu_spike_generation(
            input_pointer,
            input_dim_c0,
            input_dim_c1,
            input_dim_c2,
            random_values_pointer,
            random_values_dim_c0,
            random_values_dim_c1,
            random_values_dim_c2,
            output_pointer,
            output_dim_c0,
            output_dim_c1,
            output_dim_c2,
            x_dim,
            y_dim,
            spike_dim,
            h_dim,
            number_of_pattern);
    }

    return true;
};

bool SpikeGeneration2DManyIP::spike_generation(
    float* input_pointer,
    size_t input_dim_c0,
    size_t input_dim_c1,
    size_t input_dim_c2,
    float* random_values_pointer,
    size_t random_values_dim_c0,
    size_t random_values_dim_c1,
    size_t random_values_dim_c2,
    int64_t* output_pointer,
    size_t output_dim_c0,
    size_t output_dim_c1,
    size_t output_dim_c2,
    size_t x_dim,
    size_t y_dim,
    size_t spike_dim,
    size_t h_dim,
    size_t pattern_id)
{

    size_t counter;
    size_t counter_x = 0;
    size_t counter_y = 0;

    float* p_ptr = nullptr;
    int64_t* out_ptr = nullptr;
    float* rand_ptr = nullptr;

    for (counter_x = 0; counter_x < x_dim; counter_x++)
    {
        for (counter_y = 0; counter_y < y_dim; counter_y++)
        {
            p_ptr = input_pointer + pattern_id * input_dim_c0 +
                counter_x * input_dim_c2 + counter_y;
            // + counter * input_dim_c1

            out_ptr = output_pointer + pattern_id * output_dim_c0 +
                counter_x * output_dim_c2 + counter_y;
            // + counter * output_dim_c1

            rand_ptr = random_values_pointer +
                pattern_id * random_values_dim_c0 +
                counter_x * random_values_dim_c2 + counter_y;
            // + counter * random_values_dim_c1

            for (counter = 0; counter < spike_dim; counter++)
            {
                out_ptr[counter * output_dim_c1] = lower_bound(p_ptr,
                    h_dim,
                    input_dim_c1,
                    rand_ptr[counter * random_values_dim_c1]);
            }
        }
    }

    return true;
};

// algorithmic idea stolen from libc++
size_t SpikeGeneration2DManyIP::lower_bound(float* data_ptr,
    size_t data_length,
    size_t data_ptr_stride,
    float compare_to_value)
{

    size_t start_of_range = 0;
    size_t length_of_range = data_length;

    while (length_of_range != 0)
    {
        size_t half_length = length_of_range >> 1;
        size_t actual_position = start_of_range + half_length;

        if (data_ptr[actual_position * data_ptr_stride] < compare_to_value)
        {
            start_of_range = ++actual_position;
            length_of_range -= half_length + 1;
        }
        else
            length_of_range = half_length;
    }
    return start_of_range;
};

__device__ size_t gpu_lower_bound(float* __restrict__ data_ptr,
    size_t data_length,
    size_t data_ptr_stride,
    float compare_to_value)
{

    size_t start_of_range = 0;
    size_t length_of_range = data_length;

    while (length_of_range != 0)
    {
        size_t half_length = length_of_range >> 1;
        size_t actual_position = start_of_range + half_length;

        if (data_ptr[actual_position * data_ptr_stride] < compare_to_value)
        {
            start_of_range = ++actual_position;
            length_of_range -= half_length + 1;
        }
        else
            length_of_range = half_length;
    }
    return start_of_range;
};

__global__ void kernel_spike_generation(
    float* __restrict__ input_pointer,
    size_t input_dim_c0,
    size_t input_dim_c1,
    size_t input_dim_c2,
    float* __restrict__ random_values_pointer,
    size_t random_values_dim_c0,
    size_t random_values_dim_c1,
    size_t random_values_dim_c2,
    int64_t* __restrict__ output_pointer,
    size_t output_dim_c0,
    size_t output_dim_c1,
    size_t output_dim_c2,
    size_t x_dim,
    size_t y_dim,
    size_t spike_dim,
    size_t h_dim,
    size_t max_threadable_tasks)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < max_threadable_tasks)
    {

        size_t pattern_id = idx / spike_dim;
        size_t position_spike = idx - (pattern_id * spike_dim);

        size_t position_x = blockIdx.y;
        size_t position_y = blockIdx.z;

        float* p_ptr = input_pointer + pattern_id * input_dim_c0 +
            position_x * input_dim_c2 + position_y;

        int64_t* out_ptr = output_pointer + pattern_id * output_dim_c0 +
            position_x * output_dim_c2 + position_y
            + position_spike * output_dim_c1;

        float* rand_ptr = random_values_pointer +
            pattern_id * random_values_dim_c0 +
            position_x * random_values_dim_c2 + position_y
            + position_spike * random_values_dim_c1;

        *out_ptr = gpu_lower_bound(p_ptr,
            h_dim,
            input_dim_c1,
            *rand_ptr);
    }
};

bool SpikeGeneration2DManyIP::gpu_spike_generation(
    float* input_pointer,
    size_t input_dim_c0,
    size_t input_dim_c1,
    size_t input_dim_c2,
    float* random_values_pointer,
    size_t random_values_dim_c0,
    size_t random_values_dim_c1,
    size_t random_values_dim_c2,
    int64_t* output_pointer,
    size_t output_dim_c0,
    size_t output_dim_c1,
    size_t output_dim_c2,
    size_t x_dim,
    size_t y_dim,
    size_t spike_dim,
    size_t h_dim,
    size_t number_of_pattern)
{
    hipError_t status;
    assert((x_dim < 65535));
    assert((y_dim < 65535));

    // // //////////////////////////////////////
    // // Get infos about the device
    // // //////////////////////////////////////

    // int device;
    // hipDeviceProp_t prop;

    // status = hipGetDevice(&device);
    // assert((status == hipSuccess));
    // // std::cout << "Device ID: " << device << std::endl;

    // status = hipGetDeviceProperties(&prop, device);
    // assert((status == hipSuccess));
    // // std::cout << "Device name: " << prop.name << std::endl;

  // //////////////////////////////////////
  // Calculate the distribution on the GPU
  // //////////////////////////////////////

    int min_grid_size;
    int block_size;
    int grid_size;

    size_t dynamic_s_mem_size = 0;
    size_t max_threadable_tasks = number_of_pattern * spike_dim * x_dim * y_dim;

    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=blocksize#occupancy-calculator
    status = hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
        (void*)kernel_spike_generation,
        dynamic_s_mem_size, max_threadable_tasks);
    assert((status == hipSuccess));

    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications
    // Maximum dimensionality of grid of thread blocks: 3
    // Maximum x -dimension of a grid of thread blocks: (2^31)-1
    // Maximum y- or z-dimension of a grid of thread blocks: 65535

    // Round up according to array size
    // (I will separate x and y into other grid dimentsions soon)
    grid_size = ((number_of_pattern * spike_dim) + block_size - 1) / block_size;

    // std::cout << min_grid_size << std::endl;
    // std::cout << grid_size << std::endl;
    // std::cout << block_size << std::endl;
    // std::cout << max_threadable_tasks << std::endl;

    dim3 grid(grid_size, x_dim, y_dim);


    kernel_spike_generation<<<grid, block_size >>>(
        input_pointer,
        input_dim_c0,
        input_dim_c1,
        input_dim_c2,
        random_values_pointer,
        random_values_dim_c0,
        random_values_dim_c1,
        random_values_dim_c2,
        output_pointer,
        output_dim_c0,
        output_dim_c1,
        output_dim_c2,
        x_dim,
        y_dim,
        spike_dim,
        h_dim,
        (number_of_pattern * spike_dim));

    hipDeviceSynchronize();

    return true;
};