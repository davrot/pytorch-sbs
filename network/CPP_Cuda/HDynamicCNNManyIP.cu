#include "hip/hip_runtime.h"
#include "HDynamicCNNManyIP.h"

#include <omp.h>
#include <stdio.h>
#include <string.h>

#include <algorithm>
#include <cassert>
#include <iostream>


HDynamicCNNManyIP::HDynamicCNNManyIP()
{

};

HDynamicCNNManyIP::~HDynamicCNNManyIP()
{

};

bool HDynamicCNNManyIP::update_entrypoint(
    int64_t h_pointer_addr,
    int64_t h_dim_0,
    int64_t h_dim_1,
    int64_t h_dim_2,
    int64_t h_dim_3,
    int64_t epsilon_xy_pointer_addr,
    int64_t epsilon_xy_dim_0,
    int64_t epsilon_xy_dim_1,
    int64_t epsilon_xy_dim_2,
    int64_t epsilon_t_pointer_addr,
    int64_t epsilon_t_dim_0,
    int64_t weights_pointer_addr,
    int64_t weights_dim_0,
    int64_t weights_dim_1,
    int64_t input_pointer_addr,
    int64_t input_dim_0,
    int64_t input_dim_1,
    int64_t input_dim_2,
    int64_t input_dim_3,
    int64_t init_vector_pointer_addr,
    int64_t init_vector_dim_0,
    int64_t number_of_processes,
    float forgetting_offset,
    int64_t gpu_tuning_factor)
{

    size_t number_of_pattern = input_dim_0;

    size_t h_dim = init_vector_dim_0;
    float* h_init_ptr = (float*)init_vector_pointer_addr;
    assert((h_init_ptr != nullptr));
    assert((h_dim > 0));

    float* h_pointer = (float*)h_pointer_addr;
    assert((h_pointer != nullptr));
    assert((h_dim_0 > 0));
    assert((h_dim_1 > 0));
    assert((h_dim_2 > 0));
    assert((h_dim_3 > 0));

    size_t h_dim_c0 = h_dim_1 * h_dim_2 * h_dim_3;
    size_t h_dim_c1 = h_dim_2 * h_dim_3;
    size_t h_dim_c2 = h_dim_3;

    float* epsilon_xy_pointer = (float*)epsilon_xy_pointer_addr;
    assert((epsilon_xy_pointer != nullptr));
    assert((epsilon_xy_dim_0 > 0));
    assert((epsilon_xy_dim_1 > 0));

    size_t epsilon_xy_dim_c0 = epsilon_xy_dim_2 * epsilon_xy_dim_1;
    size_t epsilon_xy_dim_c1 = epsilon_xy_dim_2;

    float* epsilon_t_pointer = (float*)epsilon_t_pointer_addr;
    assert((epsilon_t_pointer != nullptr));
    assert((epsilon_t_dim_0 > 0));

    float* weights_pointer = (float*)weights_pointer_addr;
    assert((weights_pointer != nullptr));
    assert((weights_dim_0 > 0));
    assert((weights_dim_1 > 0));

    size_t weights_dim_c0 = weights_dim_1;

    int64_t* input_pointer = (int64_t*)input_pointer_addr;
    assert((input_pointer != nullptr));
    assert((input_dim_0 > 0));
    assert((input_dim_1 > 0));
    assert((input_dim_2 > 0));
    assert((input_dim_3 > 0));

    size_t input_dim_c0 = input_dim_1 * input_dim_2 * input_dim_3;
    size_t input_dim_c1 = input_dim_2 * input_dim_3;
    size_t input_dim_c2 = input_dim_3;

    assert((h_dim == weights_dim_1));
    size_t number_of_spikes = input_dim_1;
    size_t dim_x = input_dim_2;
    size_t dim_y = input_dim_3;

    float forgetting_offset_local = forgetting_offset / static_cast<float>(h_dim);


    // --------------------
    if (number_of_processes > 0)
    {
        omp_set_num_threads(number_of_processes);

        size_t pattern_id;
#pragma omp parallel for
        for (pattern_id = 0; pattern_id < number_of_pattern; pattern_id++)
        {
            update(
                h_init_ptr,
                h_pointer,
                h_dim_c0,
                h_dim_c1,
                h_dim_c2,
                h_dim,
                epsilon_xy_pointer,
                epsilon_xy_dim_c0,
                epsilon_xy_dim_c1,
                epsilon_t_pointer,
                weights_pointer,
                weights_dim_c0,
                input_pointer,
                input_dim_c0,
                input_dim_c1,
                input_dim_c2,
                number_of_spikes,
                dim_x,
                dim_y,
                forgetting_offset,
                forgetting_offset_local,
                pattern_id);
        }
    }
    else
    {
        gpu_update(
            h_init_ptr,
            h_pointer,
            h_dim_c0,
            h_dim_c1,
            h_dim_c2,
            h_dim,
            epsilon_xy_pointer,
            epsilon_xy_dim_c0,
            epsilon_xy_dim_c1,
            epsilon_t_pointer,
            weights_pointer,
            weights_dim_c0,
            input_pointer,
            input_dim_c0,
            input_dim_c1,
            input_dim_c2,
            number_of_spikes,
            dim_x,
            dim_y,
            forgetting_offset,
            forgetting_offset_local,
            number_of_pattern,
            gpu_tuning_factor);

    }
    return true;
};


bool HDynamicCNNManyIP::update(
    float* h_init_ptr,
    float* h_pointer,
    size_t h_dim_c0,
    size_t h_dim_c1,
    size_t h_dim_c2,
    size_t h_dim,
    float* epsilon_xy_pointer,
    size_t epsilon_xy_dim_c0,
    size_t epsilon_xy_dim_c1,
    float* epsilon_t_pointer,
    float* weights_pointer,
    size_t weights_dim_c0,
    int64_t* input_pointer,
    size_t input_dim_c0,
    size_t input_dim_c1,
    size_t input_dim_c2,
    size_t number_of_spikes,
    size_t dim_x,
    size_t dim_y,
    float forgetting_offset,
    float forgetting_offset_local,
    size_t pattern_id)
{

    float* h_ptr;
    float* epsilon_xy_ptr;
    int64_t* input_ptr;

    size_t counter_x;
    size_t counter_y;

    for (counter_x = 0; counter_x < dim_x; counter_x++)
    {
        for (counter_y = 0; counter_y < dim_y; counter_y++)
        {
            epsilon_xy_ptr = epsilon_xy_pointer +
                counter_x * epsilon_xy_dim_c1 + counter_y;

            h_ptr = h_pointer +
                pattern_id * h_dim_c0 + counter_x * h_dim_c2 + counter_y;

            input_ptr = input_pointer +
                pattern_id * input_dim_c0 + counter_x * input_dim_c2 + counter_y;

            update_one_ip(
                h_init_ptr,
                h_ptr,
                h_dim_c1,
                h_dim,
                weights_pointer,
                weights_dim_c0,
                input_ptr,
                input_dim_c1,
                epsilon_xy_ptr,
                epsilon_xy_dim_c0,
                epsilon_t_pointer,
                number_of_spikes,
                forgetting_offset,
                forgetting_offset_local);

        }
    }

    return true;
};

void HDynamicCNNManyIP::update_one_ip(
    float* h_init_ptr,
    float* h_pointer,
    size_t h_dim_c1,
    size_t h_dim,
    float* weights_pointer,
    size_t weights_dim_c0,
    int64_t* input_pointer,
    size_t input_dim_c1,
    float* epsilon_xy_pointer,
    size_t epsilon_xy_dim_c0,
    float* epsilon_t_pointer,
    size_t number_of_spikes,
    float forgetting_offset,
    float forgetting_offset_local)
{

    float* h_temp = new float[h_dim];
    float* h_subsegment = new float[h_dim];

    memcpy(h_subsegment, h_init_ptr, sizeof(float) * h_dim);

    size_t counter_spike;
    size_t counter;

    float h_temp_sum;
    float temp_value;

    float epsilon_subsegment;
    float epsilon_scale = 1.0;

    int64_t* spike;
    float* w_ptr;

    for (counter_spike = 0; counter_spike < number_of_spikes; counter_spike++)
    {
        if (epsilon_scale > 1E10)
        {
            temp_value = 1.0 / epsilon_scale;

#pragma omp simd
            for (counter = 0; counter < h_dim; counter++)
            {
                h_subsegment[counter] *= temp_value;
            }

            epsilon_scale = 1.0;
        }

        spike = input_pointer + counter_spike * input_dim_c1;

        if (*spike >= 0)
        {
            epsilon_subsegment =
                epsilon_xy_pointer[*spike *epsilon_xy_dim_c0] * epsilon_t_pointer[counter_spike];

            w_ptr = weights_pointer + *spike * weights_dim_c0;

            memcpy(h_temp, h_subsegment, sizeof(float) * h_dim);

#pragma omp simd
            for (counter = 0; counter < h_dim; counter++)
            {
                h_temp[counter] *= w_ptr[counter];
            }

            h_temp_sum = 0.0;
#pragma omp simd reduction(+ : h_temp_sum)
            for (counter = 0; counter < h_dim; counter++)
            {
                h_temp_sum += h_temp[counter];
            }

            if (h_temp_sum > 1E-10)
            {
                temp_value = epsilon_scale * epsilon_subsegment / h_temp_sum;

#pragma omp simd
                for (counter = 0; counter < h_dim; counter++)
                {
                    h_temp[counter] *= temp_value;
                }

#pragma omp simd
                for (counter = 0; counter < h_dim; counter++)
                {
                    h_subsegment[counter] += h_temp[counter];
                }

                if (forgetting_offset_local > 0.0)
                {
                    temp_value =
                        epsilon_scale * epsilon_subsegment * forgetting_offset_local;

#pragma omp simd
                    for (counter = 0; counter < h_dim; counter++)
                    {
                        h_subsegment[counter] += temp_value;
                    }

                    epsilon_scale *=
                        1.0 + epsilon_subsegment * (1.0 + forgetting_offset);
                }
                else
                {
                    epsilon_scale *= 1.0 + epsilon_subsegment * 1.0;
                }
            }
        }
    }

    temp_value = 1.0 / epsilon_scale;
#pragma omp simd
    for (counter = 0; counter < h_dim; counter++)
    {
        h_pointer[counter * h_dim_c1] =
            h_subsegment[counter] * temp_value;
    }

    delete[] h_temp;
    delete[] h_subsegment;

    return;
};

__device__ void gpu_update_one_ip(
    float* __restrict__ h_init_ptr,
    float* __restrict__ h_pointer,
    size_t h_dim_c1,
    size_t h_dim,
    float* __restrict__ weights_pointer,
    size_t weights_dim_c0,
    int64_t* input_pointer,
    size_t input_dim_c1,
    float* __restrict__ epsilon_xy_pointer,
    size_t epsilon_xy_dim_c0,
    float* __restrict__ epsilon_t_pointer,
    size_t number_of_spikes,
    float forgetting_offset,
    float forgetting_offset_local,
    float* __restrict__ h_temp,
    float* __restrict__ h_subsegment
)
{

    size_t counter_spike;
    size_t counter;

    float h_temp_sum;
    float temp_value;

    float epsilon_subsegment;
    float epsilon_scale = 1.0;

    int64_t* spike;
    float* w_ptr;

    // float* h_temp = new float[h_dim];
    // float* h_subsegment = new float[h_dim];

    // Initialize the sub-segement
    for (counter = 0; counter < h_dim; counter++)
    {
        h_subsegment[counter] = h_init_ptr[counter];
    }

    for (counter_spike = 0; counter_spike < number_of_spikes; counter_spike++)
    {
        if (epsilon_scale > 1E10)
        {
            temp_value = 1.0 / epsilon_scale;

            for (counter = 0; counter < h_dim; counter++)
            {
                h_subsegment[counter] *= temp_value;
            }

            epsilon_scale = 1.0;
        }

        spike = input_pointer + counter_spike * input_dim_c1;

        if (*spike >= 0)
        {
            epsilon_subsegment =
                epsilon_xy_pointer[*spike *epsilon_xy_dim_c0] * epsilon_t_pointer[counter_spike];

            w_ptr = weights_pointer + *spike * weights_dim_c0;

            for (counter = 0; counter < h_dim; counter++)
            {
                h_temp[counter] = h_subsegment[counter] * w_ptr[counter];
            }

            h_temp_sum = 0.0;

            for (counter = 0; counter < h_dim; counter++)
            {
                h_temp_sum += h_temp[counter];
            }

            if (h_temp_sum > 1E-10)
            {
                temp_value = epsilon_scale * epsilon_subsegment / h_temp_sum;

                for (counter = 0; counter < h_dim; counter++)
                {
                    h_temp[counter] *= temp_value;
                }

                for (counter = 0; counter < h_dim; counter++)
                {
                    h_subsegment[counter] += h_temp[counter];
                }

                if (forgetting_offset_local > 0.0)
                {
                    temp_value =
                        epsilon_scale * epsilon_subsegment * forgetting_offset_local;

                    for (counter = 0; counter < h_dim; counter++)
                    {
                        h_subsegment[counter] += temp_value;
                    }

                    epsilon_scale *=
                        1.0 + epsilon_subsegment * (1.0 + forgetting_offset);
                }
                else
                {
                    epsilon_scale *= 1.0 + epsilon_subsegment * 1.0;
                }
            }
        }
    }

    temp_value = 1.0 / epsilon_scale;

    for (counter = 0; counter < h_dim; counter++)
    {
        h_pointer[counter * h_dim_c1] =
            h_subsegment[counter] * temp_value;
    }

    // delete[] h_temp;
    // delete[] h_subsegment;

    return;
};

__global__ void kernel_spike_generation(
    float* __restrict__ h_init_ptr,
    float* __restrict__ h_pointer,
    size_t h_dim_c0,
    size_t h_dim_c1,
    size_t h_dim_c2,
    size_t h_dim,
    float* __restrict__ weights_pointer,
    size_t weights_dim_c0,
    int64_t* __restrict__ input_pointer,
    size_t input_dim_c0,
    size_t input_dim_c1,
    size_t input_dim_c2,
    float* __restrict__ epsilon_xy_pointer,
    size_t epsilon_xy_dim_c0,
    size_t epsilon_xy_dim_c1,
    float* __restrict__ epsilon_t_pointer,
    size_t number_of_spikes,
    float forgetting_offset,
    float forgetting_offset_local,
    size_t dim_x,
    size_t dim_y,
    size_t dim_xy,
    size_t max_threadable_tasks,
    float* __restrict__ temp_memory_a,
    float* __restrict__ temp_memory_b
)
{

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < max_threadable_tasks)
    {
        float* h_ptr;
        float* epsilon_xy_ptr;
        int64_t* input_ptr;

        float* temp_memory_ptr_a = temp_memory_a + idx * h_dim;
        float* temp_memory_ptr_b = temp_memory_b + idx * h_dim;

        // int pattern_id = idx; 
        int pattern_id = idx / dim_xy;
        int position_xy = idx - (pattern_id * dim_xy);

        // size_t position_x = blockIdx.y;
        // size_t position_y = blockIdx.z;
        size_t position_x = position_xy / dim_y;
        size_t position_y = position_xy - (position_x * dim_y);

        epsilon_xy_ptr = epsilon_xy_pointer +
            position_x * epsilon_xy_dim_c1 + position_y;

        h_ptr = h_pointer +
            pattern_id * h_dim_c0 + position_x * h_dim_c2 + position_y;

        input_ptr = input_pointer +
            pattern_id * input_dim_c0 + position_x * input_dim_c2 + position_y;

        gpu_update_one_ip(
            h_init_ptr,
            h_ptr,
            h_dim_c1,
            h_dim,
            weights_pointer,
            weights_dim_c0,
            input_ptr,
            input_dim_c1,
            epsilon_xy_ptr,
            epsilon_xy_dim_c0,
            epsilon_t_pointer,
            number_of_spikes,
            forgetting_offset,
            forgetting_offset_local,
            temp_memory_ptr_a,
            temp_memory_ptr_b
        );

    }

};

// Let's face it... We need a better way to paralelize it...
bool HDynamicCNNManyIP::gpu_update(
    float* h_init_ptr,
    float* h_pointer,
    size_t h_dim_c0,
    size_t h_dim_c1,
    size_t h_dim_c2,
    size_t h_dim,
    float* epsilon_xy_pointer,
    size_t epsilon_xy_dim_c0,
    size_t epsilon_xy_dim_c1,
    float* epsilon_t_pointer,
    float* weights_pointer,
    size_t weights_dim_c0,
    int64_t* input_pointer,
    size_t input_dim_c0,
    size_t input_dim_c1,
    size_t input_dim_c2,
    size_t number_of_spikes,
    size_t dim_x,
    size_t dim_y,
    float forgetting_offset,
    float forgetting_offset_local,
    size_t number_of_pattern,
    size_t gpu_tuning_factor)
{

    hipError_t status;
    assert((dim_x < 65535));
    assert((dim_y < 65535));

    // // //////////////////////////////////////
    // // Get infos about the device
    // // //////////////////////////////////////

    // int device;
    // hipDeviceProp_t prop;

    // status = hipGetDevice(&device);
    // assert((status == hipSuccess));
    // // std::cout << "Device ID: " << device << std::endl;

    // status = hipGetDeviceProperties(&prop, device);
    // assert((status == hipSuccess));
    // // std::cout << "Device name: " << prop.name << std::endl;

    // int _cuda_heap_size_in_mb = 16;
    // status = hipDeviceSetLimit(hipLimitMallocHeapSize, _cuda_heap_size_in_mb * (1 << 20));
    // assert((status == hipSuccess));

    // size_t pValue;
    // hipDeviceGetLimit(&pValue, hipLimitMallocHeapSize);
    // std::cout << pValue << " " << (pValue/(2*4*h_dim)) << std::endl;
    // exit(1);


    // //////////////////////////////////////
    // Calculate the distribution on the GPU
    // //////////////////////////////////////

    int min_grid_size;
    int block_size;
    int grid_size;

    size_t dynamic_s_mem_size = 0;
    size_t max_threadable_tasks = number_of_pattern * dim_x * dim_y;

    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=blocksize#occupancy-calculator
    status = hipOccupancyMaxPotentialBlockSize(&min_grid_size, &block_size,
        (void*)kernel_spike_generation,
        dynamic_s_mem_size, max_threadable_tasks);
    assert((status == hipSuccess));

    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications
    // Maximum dimensionality of grid of thread blocks: 3
    // Maximum x -dimension of a grid of thread blocks: (2^31)-1
    // Maximum y- or z-dimension of a grid of thread blocks: 65535

    // Reduce the automatic block size with our guess 
    if ((gpu_tuning_factor > 0) && (gpu_tuning_factor < block_size))
    {
        block_size = int(gpu_tuning_factor);
    }
    // Round up according to array size
    // (I will separate x and y into other grid dimentsions soon)
    // grid_size = (number_of_pattern + block_size - 1) / block_size;
    grid_size = (max_threadable_tasks + block_size - 1) / block_size;

    // std::cout << min_grid_size << std::endl;
    // std::cout << grid_size << std::endl;
    // std::cout << block_size << std::endl;
    // std::cout << max_threadable_tasks << std::endl;

    //dim3 grid(grid_size, dim_x, dim_y);

    float* temp_memory_a = nullptr;
    status = hipMalloc((void**)&temp_memory_a, h_dim * max_threadable_tasks * sizeof(float));
    assert((status == hipSuccess));

    float* temp_memory_b = nullptr;
    status = hipMalloc((void**)&temp_memory_b, h_dim * max_threadable_tasks * sizeof(float));
    assert((status == hipSuccess));


    //kernel_spike_generation<<<grid, block_size >>>(
    kernel_spike_generation<<<grid_size, block_size >>>(
        h_init_ptr,
        h_pointer,
        h_dim_c0,
        h_dim_c1,
        h_dim_c2,
        h_dim,
        weights_pointer,
        weights_dim_c0,
        input_pointer,
        input_dim_c0,
        input_dim_c1,
        input_dim_c2,
        epsilon_xy_pointer,
        epsilon_xy_dim_c0,
        epsilon_xy_dim_c1,
        epsilon_t_pointer,
        number_of_spikes,
        forgetting_offset,
        forgetting_offset_local,
        dim_x,
        dim_y,
        (dim_x * dim_y),
        //number_of_pattern
        max_threadable_tasks,
        temp_memory_a,
        temp_memory_b
        );

    status = hipDeviceSynchronize();
    assert((status == hipSuccess));

    status = hipFree(temp_memory_a);
    assert((status == hipSuccess));

    status = hipFree(temp_memory_b);
    assert((status == hipSuccess));


    return true;
};


void HDynamicCNNManyIP::gpu_occupancy_export(
    size_t dim_x,
    size_t dim_y,
    size_t number_of_pattern,
    size_t h_dim,
    int64_t setting_memory_addr,
    size_t setting_dim_0,
    size_t setting_dim_1)
{
    return;
};

void HDynamicCNNManyIP::gpu_occupancy_import(
    int64_t setting_memory_addr,
    size_t setting_dim_0,
    size_t setting_dim_1
)
{
    return;
};